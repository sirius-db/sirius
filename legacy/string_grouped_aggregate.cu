#include "hip/hip_runtime.h"
/*
 * Copyright 2025, Sirius Contributors.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_helper.cuh"
#include "gpu_physical_grouped_aggregate.hpp"
#include "gpu_buffer_manager.hpp"
#include "log/logging.hpp"

namespace duckdb {

struct sort_keys_type_string {
  uint64_t* keys;
  uint64_t num_key;

  __host__ __device__ sort_keys_type_string() {}
  __host__ __device__ sort_keys_type_string(uint64_t* _keys, uint64_t _num_key) : keys(_keys), num_key(_num_key) {}

  __host__ __device__ bool operator<(const sort_keys_type_string& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) {
            return keys[i] < other.keys[i];
        }
      }
      return true;
    }

    __host__ __device__ bool operator==(const sort_keys_type_string& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) return false;
      }
      return true;
    }

    __host__ __device__ bool operator!=(const sort_keys_type_string& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) return true;
      }
      return false;
    }
};

struct CustomLessString
{
  __device__ bool operator()(const sort_keys_type_string &lhs, const sort_keys_type_string &rhs) {
      for (uint64_t i = 0; i < lhs.num_key; i++) {
            if (lhs.keys[i] != rhs.keys[i]) {
                return lhs.keys[i] < rhs.keys[i];
            }
      }
      return true;
  }
};

struct CustomSumString
{
    template <typename T>
    __host__ __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

struct CustomMinString
{
    template <typename T>
    __host__ __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};

struct CustomMaxString
{
    template <typename T>
    __host__ __device__ __forceinline__
     T operator()(const T &a, const T &b) const {
        return (b > a) ? b : a;
    }
};

template <typename T, int B, int I>
__global__ void fill_offset(uint64_t* offset, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            offset[tile_offset + threadIdx.x + ITEM * B] = sizeof(T) * (tile_offset + threadIdx.x + ITEM * B);
        }
    }
}

template <int B, int I>
__global__ void columns_to_rows_string(uint8_t **a, uint8_t* result, uint64_t **input_offset, uint64_t* key_length,
            sort_keys_type_string* temp, uint64_t N, uint64_t num_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    uint64_t total_length = 0;
    for (uint64_t key = 0; key < (num_keys - 1); key ++) {
        total_length += key_length[key];
    }
    //add the row ids into the total length
    total_length += sizeof(uint64_t);

    uint64_t meta_num_keys = (total_length + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    uint64_t total_length_bytes = meta_num_keys * sizeof(uint64_t);

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t output_start_idx = offset * total_length_bytes;
            memset(result + output_start_idx, 0, total_length_bytes * sizeof(uint8_t));
            //copy the keys without the row ids
            for (uint64_t key = 0; key < (num_keys - 1); key ++) {
                uint64_t input_length = input_offset[key][offset + 1] - input_offset[key][offset];
                uint64_t input_start_idx = input_offset[key][offset];
                memcpy(result + output_start_idx, a[key] + input_start_idx, input_length * sizeof(uint8_t));
                output_start_idx += key_length[key];
            }
            //copy the row ids
            memcpy(result + (offset * total_length_bytes) + ((meta_num_keys - 1) * sizeof(uint64_t)), a[num_keys - 1] + (offset * sizeof(uint64_t)), sizeof(uint64_t));
            temp[offset] = sort_keys_type_string(reinterpret_cast<uint64_t*>(&result[offset * total_length_bytes]), meta_num_keys);
        }
    }
}

template <int B, int I>
__global__ void compact_string_offset(uint64_t* group_idx, uint64_t** group_byte_offset, uint64_t** result_offset, uint64_t N, uint64_t num_keys) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            if (offset == N - 1) {
                uint64_t out_idx = group_idx[offset];
                for (uint64_t key = 0; key < num_keys; key ++) {
                    result_offset[key][out_idx] = group_byte_offset[key][offset];
                }
            } else if ((offset < (N - 1)) && (group_idx[offset] != group_idx[offset + 1])) {
                uint64_t out_idx = group_idx[offset];
                for (uint64_t key = 0; key < num_keys; key ++) {
                    cudaAssert(group_byte_offset[key][offset] != group_byte_offset[key][offset + 1]);
                    result_offset[key][out_idx] = group_byte_offset[key][offset];
                }
            }
        }
    }
}

template <int B, int I>
__global__ void rows_to_columns_string(uint64_t* group_idx, sort_keys_type_string *row_keys, uint8_t** col_keys, uint64_t **group_byte_offset, uint64_t* key_length,
    uint64_t N, uint64_t num_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            //we should write out the offset
            if (group_idx[offset] != group_idx[offset + 1]) {
                uint64_t out_idx = group_idx[offset];
                uint64_t key_length_bytes = 0;
                for (uint64_t key = 0; key < num_keys; key ++) {
                    cudaAssert(group_byte_offset[key][offset] != group_byte_offset[key][offset + 1]);
                    uint64_t out_offset = group_byte_offset[key][offset];
                    uint64_t actual_key_length = group_byte_offset[key][offset + 1] - group_byte_offset[key][offset];
                    uint8_t* ptr = reinterpret_cast<uint8_t*>(row_keys[out_idx].keys);
                    memcpy(col_keys[key] + out_offset, ptr + key_length_bytes, actual_key_length * sizeof(uint8_t));
                    key_length_bytes += key_length[key];
                }
            }
        }
    }
}

template <int B, int I>
__global__ void get_len(uint64_t* offset, uint64_t* len, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t idx = tile_offset + threadIdx.x + ITEM * B;
            len[idx] = offset[idx + 1] - offset[idx];
        }
    }
}


template <int B, int I>
__global__ void distinct_string(uint64_t* distinct_mark, uint64_t* distinct_len, uint64_t* len, sort_keys_type_string *sort_keys, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            if (offset == 0 || (offset > 0  && (sort_keys[offset] != sort_keys[offset - 1]))) {
                distinct_mark[offset] = 1;
                distinct_len[offset] = len[offset];
            } else {
                distinct_mark[offset] = 0;
                distinct_len[offset] = 0;
            }
        }
    }
}

template <typename T, int B, int I>
__global__ void gather_and_modify(const T *a, T* result, sort_keys_type_string *sort_keys, uint64_t N, uint64_t meta_num_keys) {
    cudaAssert(meta_num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t items_ids = sort_keys[offset].keys[meta_num_keys - 1];
            result[offset] = a[items_ids];
            sort_keys[offset] = sort_keys_type_string(sort_keys[offset].keys, meta_num_keys - 1);
        }
    }
}

template <typename T, int B, int I>
__global__ void gather(const T *a, T* result, sort_keys_type_string *sort_keys, uint64_t N, uint64_t num_keys) {

    cudaAssert(num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t items_ids = sort_keys[offset].keys[num_keys - 1];
            result[offset] = a[items_ids];
        }
    }
}

template <int B, int I>
__global__ void modify(sort_keys_type_string *sort_keys, uint64_t N, uint64_t meta_num_keys) {

    cudaAssert(meta_num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            sort_keys[offset] = sort_keys_type_string(sort_keys[offset].keys, meta_num_keys - 1);
        }
    }
}

template <int B, int I>
__global__ void sequence(uint64_t* result, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            result[tile_offset + threadIdx.x + ITEM * B] = tile_offset + threadIdx.x + ITEM * B;
        }
    }
}

template <typename T, int B, int I>
__global__ void divide(T* a, uint64_t* b, T* result, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            int offset = tile_offset + threadIdx.x + ITEM * B;
            result[offset] = a[offset] / b[offset];
        }
    }
}

template <typename T, int B, int I>
__global__ void fill_n(T* a, T b, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            a[tile_offset + threadIdx.x + ITEM * B] = b;
        }
    }
}

template
__global__ void gather_and_modify<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, sort_keys_type_string* sort_keys, uint64_t N, uint64_t meta_num_keys);
template
__global__ void gather_and_modify<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, double* result, sort_keys_type_string* sort_keys, uint64_t N, uint64_t meta_num_keys);
template
__global__ void gather<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, sort_keys_type_string* sort_keys, uint64_t N, uint64_t num_keys);

template <typename V>
void groupedStringAggregate(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode) {
    CHECK_ERROR();
    if (N == 0) {
        count[0] = 0;
        SIRIUS_LOG_DEBUG("Input size is 0");
        return;
    }

    SIRIUS_LOG_DEBUG("Launching String Grouped Aggregate Kernel");

    SETUP_TIMING();
    START_TIMER();
    
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    void     *d_temp_storage = nullptr;
    size_t   temp_storage_bytes = 0;

    //cubmax
    // Get the maximum key length for each key
    uint64_t* key_length = gpuBufferManager->customCudaMalloc<uint64_t>(num_keys, 0, 0); // store the maximum length of each key
    uint64_t** len = gpuBufferManager->customCudaHostAlloc<uint64_t*>(num_keys);
    uint64_t* original_bytes = gpuBufferManager->customCudaHostAlloc<uint64_t>(num_keys);
    for (int key = 0; key < num_keys; key++) {
        len[key] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);

        if (offset[key] == nullptr) {
            offset[key] = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
            fill_offset<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD), BLOCK_THREADS>>>(offset[key], N+1);
            CHECK_ERROR();
        }
        hipMemcpy(original_bytes + key, offset[key] + N, sizeof(uint64_t), hipMemcpyDeviceToHost);

        get_len<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD), BLOCK_THREADS>>>(offset[key], len[key], N);
        CHECK_ERROR();
        d_temp_storage = nullptr;
        temp_storage_bytes = 0;

        if (offset[key] == nullptr) {
            hipMemcpy(key_length + key, len[key], sizeof(uint64_t), hipMemcpyDeviceToDevice);
        } else {
            hipcub::DeviceReduce::Max(
            d_temp_storage, temp_storage_bytes, len[key], key_length + key, N);

            // Allocate temporary storage
            d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

            // Run min-reduction
            hipcub::DeviceReduce::Max(
            d_temp_storage, temp_storage_bytes, len[key], key_length + key, N);
            gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);
        }
    }

    uint64_t* h_key_length = gpuBufferManager->customCudaHostAlloc<uint64_t>(num_keys);
    hipMemcpy(h_key_length, key_length, num_keys * sizeof(uint64_t), hipMemcpyDeviceToHost);
    CHECK_ERROR();

    uint64_t row_id_size = sizeof(uint64_t);
    uint64_t total_length = 0;
    for (uint64_t key = 0; key < num_keys; key ++) {
        total_length += h_key_length[key];
    }
    //add the row ids into the total length
    total_length += row_id_size;
    uint64_t meta_num_keys = (total_length + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    uint64_t total_length_bytes = meta_num_keys * sizeof(uint64_t);

    //allocate temp memory and copying keys
    uint8_t* row_keys = gpuBufferManager->customCudaMalloc<uint8_t>((total_length_bytes) * N, 0, 0);
    sort_keys_type_string* materialized_temp = reinterpret_cast<sort_keys_type_string*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0));

    uint8_t** keys_row_id = gpuBufferManager->customCudaHostAlloc<uint8_t*>(num_keys + 1);
    for (uint64_t i = 0; i < num_keys; i++) {
        keys_row_id[i] = keys[i];
    }

    //generate sequence
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    uint64_t* row_sequence = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
    sequence<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(row_sequence, N);
    keys_row_id[num_keys] = reinterpret_cast<uint8_t*> (row_sequence);

    uint8_t** keys_dev = gpuBufferManager->customCudaMalloc<uint8_t*>(num_keys + 1, 0, 0);
    hipMemcpy(keys_dev, keys_row_id, (num_keys + 1) * sizeof(uint8_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    uint64_t** offset_dev = gpuBufferManager->customCudaMalloc<uint64_t*>(num_keys, 0, 0);
    hipMemcpy(offset_dev, offset, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    columns_to_rows_string<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, row_keys, offset_dev, key_length,
            materialized_temp, N, num_keys + 1);
    CHECK_ERROR();

    //perform sort-based groupby
    // Determine temporary device storage requirements
    CustomLessString custom_less;
    d_temp_storage = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    // Allocate temporary storage
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

    // Run sorting operation
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);

    SIRIUS_LOG_DEBUG("Gathering offset");
    uint64_t** group_byte_offset = gpuBufferManager->customCudaHostAlloc<uint64_t*>(num_keys);
    uint64_t* distinct_bound = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
    uint64_t* group_idx = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
    uint64_t* d_num_bytes = gpuBufferManager->customCudaMalloc<uint64_t>(num_keys, 0, 0);

    for (uint64_t key = 0; key < num_keys; key++) {
        uint64_t* temp = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
        group_byte_offset[key] = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
        hipMemset(group_byte_offset[key] + N, 0, sizeof(uint64_t));

        gather_and_modify<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(len[key], temp, materialized_temp, N, meta_num_keys);
        CHECK_ERROR();
        distinct_string<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(distinct_bound, temp, temp, materialized_temp, N);
        CHECK_ERROR();
        //cub scan
        d_temp_storage = nullptr;
        temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp, group_byte_offset[key], N + 1);

        // Allocate temporary storage for exclusive prefix sum
        d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

        // Run exclusive prefix sum
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp, group_byte_offset[key], N + 1);
        CHECK_ERROR();
        gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);

        hipMemcpy(d_num_bytes + key, group_byte_offset[key] + N, sizeof(uint64_t), hipMemcpyDeviceToDevice);
        gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(temp), 0);
        CHECK_ERROR();
    }

    //copy num_bytes over
    hipMemcpy(num_bytes, d_num_bytes, num_keys * sizeof(uint64_t), hipMemcpyDeviceToHost);
    uint64_t** group_byte_offset_dev = gpuBufferManager->customCudaMalloc<uint64_t*>(num_keys, 0, 0);;
    hipMemcpy(group_byte_offset_dev, group_byte_offset, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

    //cub scan
    d_temp_storage = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, distinct_bound, group_idx, N + 1);

    // Allocate temporary storage for exclusive prefix sum
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, distinct_bound, group_idx, N + 1);
    CHECK_ERROR();
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);

    //gather the aggregates based on the row_sequence
    SIRIUS_LOG_DEBUG("Gathering Aggregates");
    V** aggregate_keys_temp = gpuBufferManager->customCudaHostAlloc<V*>(num_aggregates);
    uint64_t** aggregate_star_temp = gpuBufferManager->customCudaHostAlloc<uint64_t*>(num_aggregates);
    sort_keys_type_string* group_by_rows = reinterpret_cast<sort_keys_type_string*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0));
    uint64_t* d_num_runs_out = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
    uint8_t** output_agg = gpuBufferManager->customCudaHostAlloc<uint8_t*>(num_aggregates);
    uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);

    for (int agg = 0; agg < num_aggregates; agg++) {
        SIRIUS_LOG_DEBUG("Aggregating {}", agg);
        hipMemset(d_num_runs_out, 0, sizeof(uint64_t));
        if (agg_mode[agg] == 4 || agg_mode[agg] == 5) { //count_star or count(null) or sum(null)
            aggregate_star_temp[agg] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
            if (agg_mode[agg] == 4) {
                fill_n<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(aggregate_star_temp[agg], 1, N);
            } else if (agg_mode[agg] == 5) {
                hipMemset(aggregate_star_temp[agg], 0, N * sizeof(double));
            }

            modify<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(materialized_temp, N, meta_num_keys);
            CHECK_ERROR();

            //perform reduce_by_key
            uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
            hipMemset(agg_star_out, 0, N * sizeof(uint64_t));

            SIRIUS_LOG_DEBUG("Reduce by key count_star");
            // Determine temporary device storage requirements
            d_temp_storage = nullptr;
            temp_storage_bytes = 0;
            CustomSumString custom_sum;
            hipcub::DeviceReduce::ReduceByKey(
                d_temp_storage, temp_storage_bytes,
                materialized_temp, group_by_rows, aggregate_star_temp[agg],
                agg_star_out, d_num_runs_out, custom_sum, N);

            CHECK_ERROR();

            // Allocate temporary storage
            d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

            // Run reduce-by-key
            hipcub::DeviceReduce::ReduceByKey(
                d_temp_storage, temp_storage_bytes,
                materialized_temp, group_by_rows, aggregate_star_temp[agg],
                agg_star_out, d_num_runs_out, custom_sum, N);

            CHECK_ERROR();

            hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
            gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_star_temp[agg]), 0);
            gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);
            count[0] = h_count[0];

            SIRIUS_LOG_DEBUG("Count: {}", count[0]);

            CHECK_ERROR();
            output_agg[agg] = reinterpret_cast<uint8_t*> (agg_star_out);
        } else {
            aggregate_keys_temp[agg] = gpuBufferManager->customCudaMalloc<V>(N, 0, 0);
            V* temp = reinterpret_cast<V*> (aggregate_keys[agg]);
            gather_and_modify<V, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(temp, aggregate_keys_temp[agg], materialized_temp, N, meta_num_keys);
            CHECK_ERROR();

            V* agg_out = gpuBufferManager->customCudaMalloc<V>(N, 0, 0);
            hipMemset(agg_out, 0, N * sizeof(V));

            CHECK_ERROR();
            if (agg_mode[agg] == 0) {
                SIRIUS_LOG_DEBUG("Reduce by key sum");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomSumString custom_sum;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_keys_temp[agg]), 0);
                output_agg[agg] = reinterpret_cast<uint8_t*> (agg_out);
                SIRIUS_LOG_DEBUG("Count: {}", count[0]);
            } else if (agg_mode[agg] == 1) {
                //Currently typename V has to be a double
                SIRIUS_LOG_DEBUG("Reduce by key avg");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomSumString custom_sum;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);

                aggregate_star_temp[agg] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
                fill_n<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(aggregate_star_temp[agg], 1, N);

                uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
                hipMemset(agg_star_out, 0, N * sizeof(uint64_t));
                hipMemset(d_num_runs_out, 0, sizeof(uint64_t));

                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_star_temp[agg],
                    agg_star_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_star_temp[agg],
                    agg_star_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                V* output = gpuBufferManager->customCudaMalloc<V>(count[0], 0, 0);
                divide<V, BLOCK_THREADS, ITEMS_PER_THREAD><<<(count[0] + tile_items - 1)/tile_items, BLOCK_THREADS>>>(agg_out, agg_star_out, output, count[0]);

                CHECK_ERROR();
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_keys_temp[agg]), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_star_temp[agg]), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(agg_star_out), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(agg_out), 0);
                output_agg[agg] = reinterpret_cast<uint8_t*> (output);
            } else if (agg_mode[agg] == 2) {
                SIRIUS_LOG_DEBUG("Reduce by key max");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomMaxString custom_max;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_max, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_max, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_keys_temp[agg]), 0);
                output_agg[agg] = reinterpret_cast<uint8_t*> (agg_out);
            } else if (agg_mode[agg] == 3) {
                SIRIUS_LOG_DEBUG("Reduce by key min");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomMinString custom_min;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_min, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_min, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_temp_storage), 0);
                gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_keys_temp[agg]), 0);
                output_agg[agg] = reinterpret_cast<uint8_t*> (agg_out);
            }
        }
    }

    uint64_t** offset_dev_result = gpuBufferManager->customCudaMalloc<uint64_t*>(num_keys, 0, 0);
    for (uint64_t i = 0; i < num_keys; i++) {
        gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(offset[i]), 0);
        offset[i] = gpuBufferManager->customCudaMalloc<uint64_t>(count[0], 0, 0);
    }
    hipMemcpy(offset_dev_result, offset, num_keys * sizeof(uint8_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    compact_string_offset<BLOCK_THREADS, ITEMS_PER_THREAD><<<((N + 1) + tile_items - 1)/tile_items, BLOCK_THREADS>>>(
            group_idx, group_byte_offset_dev, offset_dev_result, N + 1, num_keys);

    CHECK_ERROR();

    uint8_t** keys_dev_result = gpuBufferManager->customCudaMalloc<uint8_t*>(num_keys, 0, 0);
    for (uint64_t i = 0; i < num_keys; i++) {
        uint64_t* temp_num_bytes = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
        hipMemcpy(temp_num_bytes, offset[i] + count[0], sizeof(uint64_t), hipMemcpyDeviceToHost);
        gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(keys[i]), 0);
        keys[i] = gpuBufferManager->customCudaMalloc<uint8_t>(temp_num_bytes[0], 0, 0);
    }
    hipMemcpy(keys_dev_result, keys, num_keys * sizeof(uint8_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    rows_to_columns_string<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(
            group_idx, group_by_rows, keys_dev_result, group_byte_offset_dev, key_length, N, num_keys);

    CHECK_ERROR();

    for (int agg = 0; agg < num_aggregates; agg++) {
        if (agg_mode[agg] >= 0 && agg_mode[agg] <= 3) {
            gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(aggregate_keys[agg]), 0);
            aggregate_keys[agg] = output_agg[agg];
        } else {
            aggregate_keys[agg] = output_agg[agg];
        }
    }

    for (uint64_t i = 0; i < num_keys; i++) {
        gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(len[i]), 0);
        gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(group_byte_offset[i]), 0);
    }

    //free row_keys, row_sequence, materialized_temp
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(keys_dev), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(offset_dev), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(group_byte_offset_dev), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(offset_dev_result), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(keys_dev_result), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(row_keys), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(row_sequence), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(materialized_temp), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(group_by_rows), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_num_runs_out), 0); 
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(key_length), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(distinct_bound), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(group_idx), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_num_bytes), 0);
    hipDeviceSynchronize();
    SIRIUS_LOG_DEBUG("String Grouped Aggregate Count: {}\n", count[0]);

    STOP_TIMER();
}

// __global__ void add_offset(uint64_t* a, uint64_t* b, uint64_t offset, uint64_t N) {
//     uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < N) {
//         a[idx] = b[idx] + offset;
//     }
// }

// void combineStrings(uint8_t* a, uint8_t* b, uint8_t*& c, 
//         uint64_t* offset_a, uint64_t* offset_b, uint64_t*& offset_c, 
//         uint64_t num_bytes_a, uint64_t num_bytes_b, uint64_t N_a, uint64_t N_b) {
//     CHECK_ERROR();
//     if (N_a == 0 || N_b == 0) {
//         SIRIUS_LOG_DEBUG("Input size is 0");
//         return;
//     }
//     GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
//     c = gpuBufferManager->customCudaMalloc<uint8_t>(num_bytes_a + num_bytes_b, 0, 0);
//     offset_c = gpuBufferManager->customCudaMalloc<uint64_t>(N_a + N_b + 1, 0, 0);
//     hipMemcpy(c, a, num_bytes_a * sizeof(uint8_t), hipMemcpyDeviceToDevice);
//     hipMemcpy(c + num_bytes_a, b, num_bytes_b * sizeof(uint8_t), hipMemcpyDeviceToDevice);

//     hipMemcpy(offset_c, offset_a, N_a * sizeof(uint64_t), hipMemcpyDeviceToDevice);
//     add_offset<<<((N_b + 1) + BLOCK_THREADS - 1)/(BLOCK_THREADS), BLOCK_THREADS>>>(offset_c + N_a, offset_b, num_bytes_a, N_b + 1);
//     CHECK_ERROR();
//     hipDeviceSynchronize();
// }

template
void groupedStringAggregate<double>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

template
void groupedStringAggregate<uint64_t>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

}