#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_columns.hpp"

namespace duckdb {

template <typename T, int B, int I>
__global__ void materialize_expression(const T *a, T* result, uint64_t *row_ids, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            int items_ids = row_ids[tile_offset + threadIdx.x + ITEM * B];
            // if (N == 3793296 && (items_ids < 0 || items_ids >= 3793296)) printf("items_ids: %d\n", items_ids);
            result[tile_offset + threadIdx.x + ITEM * B] = a[items_ids];
            // cudaAssert(a[items_ids] == 19940101);
            // printf("Result: %ld\n", result[tile_offset + threadIdx.x + ITEM * B]);
        }
    }

}

template <typename T>
__global__ void testprintmat(T* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (uint64_t i = 0; i < N; i++) {
            printf("%lu ", a[i]);
        }
        printf("\n");
    }
}

__global__ void materialize_offset(uint64_t* offset, uint64_t* result_length, uint64_t* row_ids, size_t N) {
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < N) {
        uint64_t copy_row_id = row_ids[tid];
        uint64_t new_length = offset[copy_row_id + 1] - offset[copy_row_id];
        result_length[tid] = new_length;
        // printf("%ld %ld\n", tid, result_length[tid]);
        // printf("SET MATERALIZE: Copy Row Id - %d, New Len - %d, New Offset - %d\n", copy_row_id, new_length, materalized_offsets[tid]);
    }
}

__global__ void materialize_string(uint8_t* data, uint8_t* result, uint64_t* input_offset, uint64_t* materialized_offset, uint64_t* row_ids, size_t num_rows) {
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < num_rows) {
        uint64_t copy_row_id = row_ids[tid];
        uint64_t input_start_idx = input_offset[copy_row_id];
        uint64_t input_length = input_offset[copy_row_id + 1] - input_offset[copy_row_id];
        uint64_t output_start_idx = materialized_offset[tid];
        // printf("CHARS COPY: Copy Row Id - %ld, Src Start Idx - %ld, Src Length - %ld, Dst Write Idx - %ld\n", copy_row_id, input_start_idx, input_length, output_start_idx);
        memcpy(result + output_start_idx, data + input_start_idx, input_length * sizeof(uint8_t));
    }
}

template
__global__ void materialize_expression<int, BLOCK_THREADS, ITEMS_PER_THREAD>(const int *a, int* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<float, BLOCK_THREADS, ITEMS_PER_THREAD>(const float *a, float* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, double* result, uint64_t *row_ids, uint64_t N);
template
__global__ void materialize_expression<uint8_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint8_t *a, uint8_t* result, uint64_t *row_ids, uint64_t N);

template
__global__ void testprintmat<uint64_t>(uint64_t* a, uint64_t N);
template
__global__ void testprintmat<double>(double* a, uint64_t N);
template
__global__ void testprintmat<int>(int* a, uint64_t N);
template
__global__ void testprintmat<float>(float* a, uint64_t N);
template
__global__ void testprintmat<uint8_t>(uint8_t* a, uint64_t N);

template <typename T>
void materializeExpression(T *a, T* result, uint64_t *row_ids, uint64_t N) {
    CHECK_ERROR();
    if (N == 0) {
        printf("N is 0\n");
        return;
    }
    printf("Launching Materialize Kernel\n");
    printf("N: %lu\n", N);
    // testprintmat<T><<<1, 1>>>(a, N);
    // CHECK_ERROR();
    // testprintmat<uint64_t><<<1, 1>>>(row_ids, N);
    // CHECK_ERROR();
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    materialize_expression<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(a, result, row_ids, N);
    CHECK_ERROR();
    // thrust::device_vector<T> sorted(result, result + N);
    // thrust::sort(thrust::device, sorted.begin(), sorted.end());
    // T* raw_sorted = thrust::raw_pointer_cast(sorted.data());
    // hipMemcpy(result, raw_sorted, N * sizeof(T), hipMemcpyDeviceToDevice);
    // test<T><<<1, 1>>>(a, N);
    hipDeviceSynchronize();
}

void materializeString(uint8_t* data, uint64_t* offset, uint8_t* &result, uint64_t* &result_offset, uint64_t* row_ids, uint64_t* &new_num_bytes, uint64_t N) {

    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    //allocate temp memory and copying keys
    uint64_t* temp_len = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
    result_offset = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);

    hipMemset(temp_len + N, 0, sizeof(uint64_t));
    CHECK_ERROR();

    // Copy over the offsets
    uint64_t num_blocks = std::max((uint64_t) 1, (uint64_t) (N + BLOCK_THREADS - 1)/BLOCK_THREADS);
    materialize_offset<<<num_blocks, BLOCK_THREADS>>>(offset, temp_len, row_ids, N);
    hipDeviceSynchronize();
    CHECK_ERROR();

    //cub scan
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp_len, result_offset, N + 1);

    // Allocate temporary storage for exclusive prefix sum
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp_len, result_offset, N + 1);
    CHECK_ERROR();

    // testprintmat<uint64_t><<<1, 1>>>(result_offset, N + 1);

    new_num_bytes = new uint64_t[1];
    hipMemcpy(new_num_bytes, result_offset + N, sizeof(uint64_t), hipMemcpyDeviceToHost);
    // std::cout << "Got new chars len of " << new_num_bytes[0] << std::endl;

    CHECK_ERROR();

    result = gpuBufferManager->customCudaMalloc<uint8_t>(new_num_bytes[0], 0, 0);

    materialize_string<<<num_blocks, BLOCK_THREADS>>>(data, result, offset, result_offset, row_ids, N);
    hipDeviceSynchronize();
    CHECK_ERROR();
}

template
void materializeExpression<int>(int *a, int* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<uint64_t>(uint64_t *a, uint64_t* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<float>(float *a, float* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<double>(double *a, double* result, uint64_t *row_ids, uint64_t N);
template
void materializeExpression<uint8_t>(uint8_t *a, uint8_t* result, uint64_t *row_ids, uint64_t N);

} // namespace duckdb