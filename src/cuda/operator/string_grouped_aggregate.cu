#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_physical_grouped_aggregate.hpp"
#include "gpu_buffer_manager.hpp"

#include <chrono>

namespace duckdb {

using std::chrono::high_resolution_clock;
using std::chrono::duration;

struct sort_keys_type_string {
  uint64_t row_id;  
  uint64_t* keys;
  uint64_t num_key;

  __host__ __device__ sort_keys_type_string() {}
  __host__ __device__ sort_keys_type_string(uint64_t _row_id, uint64_t* _keys, uint64_t _num_key) : row_id(_row_id), keys(_keys), num_key(_num_key) {}

  __host__ __device__ bool operator<(const sort_keys_type_string& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) {
            return keys[i] < other.keys[i];
        }
      }
      return true;
    }

    __host__ __device__ bool operator==(const sort_keys_type_string& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) return false;
      }
      return true;
    }

    __host__ __device__ bool operator!=(const sort_keys_type_string& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) return true;
      }
      return false;
    }
};

constexpr bool V1_LOG_MODE = false;
__device__ int d_comparator_keys_compared_v1 = 0;
__device__ int d_comparator_num_comparsions_v1 = 0;

struct CustomLessString
{
 __host__ __device__ CustomLessString() {}

  __device__ bool operator()(const sort_keys_type_string &lhs, const sort_keys_type_string &rhs) {
    if constexpr(V1_LOG_MODE) {
        atomicAdd(&d_comparator_num_comparsions_v1, (int) 1);
    }
    
    for (uint64_t i = 0; i < lhs.num_key; i++) {
        if (lhs.keys[i] != rhs.keys[i]) {
            if constexpr(V1_LOG_MODE) {
                atomicAdd(&d_comparator_keys_compared_v1, (int) i);
            }
            return lhs.keys[i] < rhs.keys[i];
        }
    }

    if constexpr(V1_LOG_MODE) {
        atomicAdd(&d_comparator_keys_compared_v1, (int) lhs.num_key);
    }
    return true;
  }
};

struct CustomSumString
{
    template <typename T>
    __host__ __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return a + b;
    }
};

struct CustomMinString
{
    template <typename T>
    __host__ __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};

struct CustomMaxString
{
    template <typename T>
    __host__ __device__ __forceinline__
     T operator()(const T &a, const T &b) const {
        return (b > a) ? b : a;
    }
};

template <typename T, int B, int I>
__global__ void fill_offset(uint64_t* offset, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            offset[tile_offset + threadIdx.x + ITEM * B] = sizeof(T) * (tile_offset + threadIdx.x + ITEM * B);
        }
    }
}

template <int B, int I>
__global__ void columns_to_rows_string(uint8_t **a, uint8_t* result, uint64_t **input_offset, uint64_t* key_length,
            sort_keys_type_string* temp, uint64_t N, uint64_t num_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    uint64_t total_length = 0;
    for (uint64_t key = 0; key < (num_keys - 1); key ++) {
        total_length += key_length[key];
    }
    //add the row ids into the total length
    total_length += sizeof(uint64_t);

    uint64_t meta_num_keys = (total_length + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    uint64_t total_length_bytes = meta_num_keys * sizeof(uint64_t);

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t output_start_idx = offset * total_length_bytes;
            memset(result + output_start_idx, 0, total_length_bytes * sizeof(uint8_t));
            //copy the keys without the row ids
            for (uint64_t key = 0; key < (num_keys - 1); key ++) {
                uint64_t input_length = input_offset[key][offset + 1] - input_offset[key][offset];
                uint64_t input_start_idx = input_offset[key][offset];
                memcpy(result + output_start_idx, a[key] + input_start_idx, input_length * sizeof(uint8_t));
                output_start_idx += key_length[key];
            }
            //copy the row ids
            memcpy(result + (offset * total_length_bytes) + ((meta_num_keys - 1) * sizeof(uint64_t)), a[num_keys - 1] + (offset * sizeof(uint64_t)), sizeof(uint64_t));
            temp[offset] = sort_keys_type_string(offset, reinterpret_cast<uint64_t*>(&result[offset * total_length_bytes]), meta_num_keys);
        }
    }
}

template <int B, int I>
__global__ void compact_string_offset(uint64_t* group_idx, uint64_t** group_byte_offset, uint64_t** result_offset, uint64_t N, uint64_t num_keys) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            if (offset == N - 1) {
                uint64_t out_idx = group_idx[offset];
                for (uint64_t key = 0; key < num_keys; key ++) {
                    result_offset[key][out_idx] = group_byte_offset[key][offset];
                }
            } else if ((offset < (N - 1)) && (group_idx[offset] != group_idx[offset + 1])) {
                uint64_t out_idx = group_idx[offset];
                for (uint64_t key = 0; key < num_keys; key ++) {
                    cudaAssert(group_byte_offset[key][offset] != group_byte_offset[key][offset + 1]);
                    result_offset[key][out_idx] = group_byte_offset[key][offset];
                }
            }
        }
    }
}

__global__ void print_sort_metadata_v1() {
    float average_compare_values = (1.0 * d_comparator_keys_compared_v1)/d_comparator_num_comparsions_v1;
    printf("STRING GROUP BY V1: Performed %d row comparsions checking an average of %f values\n", d_comparator_num_comparsions_v1, average_compare_values);
}

template <int B, int I>
__global__ void rows_to_columns_string(uint64_t* group_idx, sort_keys_type_string *row_keys, uint8_t** col_keys, uint64_t **group_byte_offset, uint64_t* key_length,
    uint64_t N, uint64_t num_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            //we should write out the offset
            if (group_idx[offset] != group_idx[offset + 1]) {
                uint64_t out_idx = group_idx[offset];
                uint64_t key_length_bytes = 0;
                for (uint64_t key = 0; key < num_keys; key ++) {
                    cudaAssert(group_byte_offset[key][offset] != group_byte_offset[key][offset + 1]);
                    uint64_t out_offset = group_byte_offset[key][offset];
                    uint64_t actual_key_length = group_byte_offset[key][offset + 1] - group_byte_offset[key][offset];
                    uint8_t* ptr = reinterpret_cast<uint8_t*>(row_keys[out_idx].keys);
                    memcpy(col_keys[key] + out_offset, ptr + key_length_bytes, actual_key_length * sizeof(uint8_t));
                    key_length_bytes += key_length[key];
                }
                // char temp1[5];
                // char temp2[18];
                // memcpy(temp1, col_keys[0] + group_byte_offset[0][offset], 5);
                // memcpy(temp2, col_keys[1] + group_byte_offset[1][offset], 18);
                // printf("String %s %s\n", temp1, temp2);
                // printf("%ld %ld\n", row_keys[out_idx].keys, row_keys[out_idx].keys);
            }
        }
    }
}

template <int B, int I>
__global__ void get_len(uint64_t* offset, uint64_t* len, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t idx = tile_offset + threadIdx.x + ITEM * B;
            len[idx] = offset[idx + 1] - offset[idx];
        }
    }
}


template <int B, int I>
__global__ void distinct_string(uint64_t* distinct_mark, uint64_t* distinct_len, uint64_t* len, sort_keys_type_string *sort_keys, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            if (offset == 0 || (offset > 0  && (sort_keys[offset] != sort_keys[offset - 1]))) {
                distinct_mark[offset] = 1;
                distinct_len[offset] = len[offset];
            } else {
                distinct_mark[offset] = 0;
                distinct_len[offset] = 0;
            }
        }
    }
}

template <typename T, int B, int I>
__global__ void gather_and_modify(const T *a, T* result, sort_keys_type_string *sort_keys, uint64_t N, uint64_t meta_num_keys) {
    cudaAssert(meta_num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t items_ids = sort_keys[offset].keys[meta_num_keys - 1];
            result[offset] = a[items_ids];
            sort_keys[offset] = sort_keys_type_string(offset, sort_keys[offset].keys, meta_num_keys - 1);
        }
    }
}

template <typename T, int B, int I>
__global__ void gather(const T *a, T* result, sort_keys_type_string *sort_keys, uint64_t N, uint64_t num_keys) {

    cudaAssert(num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t items_ids = sort_keys[offset].keys[num_keys - 1];
            result[offset] = a[items_ids];
        }
    }
}

template <int B, int I>
__global__ void modify(sort_keys_type_string *sort_keys, uint64_t N, uint64_t meta_num_keys) {

    cudaAssert(meta_num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            sort_keys[offset] = sort_keys_type_string(offset, sort_keys[offset].keys, meta_num_keys - 1);
        }
    }
}

template <int B, int I>
__global__ void sequence(uint64_t* result, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            result[tile_offset + threadIdx.x + ITEM * B] = tile_offset + threadIdx.x + ITEM * B;
        }
    }
}

template <typename T, int B, int I>
__global__ void divide(T* a, uint64_t* b, T* result, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            int offset = tile_offset + threadIdx.x + ITEM * B;
            result[offset] = a[offset] / b[offset];
        }
    }
}

template <typename T, int B, int I>
__global__ void fill_n(T* a, T b, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            a[tile_offset + threadIdx.x + ITEM * B] = b;
        }
    }
}

template <typename T>
__global__ void testprint(T* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (uint64_t i = 0; i < N; i++) {
            printf("%.2f ", a[i]);
        }
        printf("\n");
    }
}

template
__global__ void gather_and_modify<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, sort_keys_type_string* sort_keys, uint64_t N, uint64_t meta_num_keys);
template
__global__ void gather_and_modify<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, double* result, sort_keys_type_string* sort_keys, uint64_t N, uint64_t meta_num_keys);
template
__global__ void gather<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, sort_keys_type_string* sort_keys, uint64_t N, uint64_t num_keys);

template <typename V>
void groupedStringAggregate(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode) {
    CHECK_ERROR();
    if (N == 0) {
        count[0] = 0;
        printf("N is 0\n");
        return;
    }

    printf("Launching String Grouped Aggregate Kernel\n");
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    void     *d_temp_storage = nullptr;
    size_t   temp_storage_bytes = 0;

    //cubmax
    auto preprocess_start_time = high_resolution_clock::now();
    // Get the maximum key length for each key
    uint64_t* key_length = gpuBufferManager->customCudaMalloc<uint64_t>(num_keys, 0, 0); // store the maximum length of each key
    uint64_t** len = new uint64_t*[num_keys];
    for (int key = 0; key < num_keys; key++) {
        len[key] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);

        if (offset[key] == nullptr) {
            offset[key] = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
            fill_offset<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD), BLOCK_THREADS>>>(offset[key], N+1);
            CHECK_ERROR();
        }

        get_len<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD), BLOCK_THREADS>>>(offset[key], len[key], N);
        CHECK_ERROR();
        d_temp_storage = nullptr;
        temp_storage_bytes = 0;

        if (offset[key] == nullptr) {
            hipMemcpy(key_length + key, len[key], sizeof(uint64_t), hipMemcpyDeviceToDevice);
        } else {
            hipcub::DeviceReduce::Max(
            d_temp_storage, temp_storage_bytes, len[key], key_length + key, N);

            // Allocate temporary storage
            d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

            // Run min-reduction
            hipcub::DeviceReduce::Max(
            d_temp_storage, temp_storage_bytes, len[key], key_length + key, N);
        }
    }

    uint64_t* h_key_length = new uint64_t[num_keys];
    hipMemcpy(h_key_length, key_length, num_keys * sizeof(uint64_t), hipMemcpyDeviceToHost);
    CHECK_ERROR();

    uint64_t row_id_size = sizeof(uint64_t);
    uint64_t total_length = 0;
    for (uint64_t key = 0; key < num_keys; key ++) {
        total_length += h_key_length[key];
    }
    //add the row ids into the total length
    total_length += row_id_size;
    uint64_t meta_num_keys = (total_length + sizeof(uint64_t) - 1) / sizeof(uint64_t);
    uint64_t total_length_bytes = meta_num_keys * sizeof(uint64_t);
    // printf("Total Length: %lu\n", total_length);
    // printf("Total Length Bytes: %lu\n", total_length_bytes);

    //allocate temp memory and copying keys
    uint64_t total_preprocessing_bytes = total_length_bytes * N;
    uint8_t* row_keys = gpuBufferManager->customCudaMalloc<uint8_t>(total_preprocessing_bytes, 0, 0);
    sort_keys_type_string* materialized_temp = reinterpret_cast<sort_keys_type_string*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0));

    uint8_t** keys_row_id = new uint8_t*[num_keys + 1];
    for (uint64_t i = 0; i < num_keys; i++) {
        keys_row_id[i] = keys[i];
    }

    //generate sequence
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    uint64_t* row_sequence = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
    sequence<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(row_sequence, N);
    keys_row_id[num_keys] = reinterpret_cast<uint8_t*> (row_sequence);

    uint8_t** keys_dev;
    hipMalloc((void**) &keys_dev, (num_keys + 1) * sizeof(uint8_t*));
    hipMemcpy(keys_dev, keys_row_id, (num_keys + 1) * sizeof(uint8_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    uint64_t** offset_dev;
    hipMalloc((void**) &offset_dev, num_keys * sizeof(uint64_t*));
    hipMemcpy(offset_dev, offset, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    columns_to_rows_string<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, row_keys, offset_dev, key_length,
            materialized_temp, N, num_keys + 1);
    hipDeviceSynchronize();
    CHECK_ERROR();
    auto preprocess_end_time = high_resolution_clock::now();
    auto preprocess_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(preprocess_end_time - preprocess_start_time).count();
    std::cout << "STRING GROUP BY V1: Preprocessing requires " << meta_num_keys << " ints per row with " << N << " rows taking " << total_preprocessing_bytes << " bytes" << std::endl;
    std::cout << "STRING GROUP BY V1: Preprocessing took " << preprocess_time_ms << " ms" << std::endl;

    //perform sort-based groupby
    // Determine temporary device storage requirements
    auto sort_start_time = high_resolution_clock::now();
    CustomLessString custom_less;
    d_temp_storage = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    // Allocate temporary storage
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

    // Run sorting operation
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    hipDeviceSynchronize();
    CHECK_ERROR();
    auto sort_end_time = high_resolution_clock::now();
    auto sort_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(sort_end_time - sort_start_time).count();
    
    // Log the results
    print_sort_metadata_v1<<<1, 1>>>();
    std::cout << "STRING GROUP BY V1: Sorting took " << sort_time_ms << " ms" << std::endl;

    auto group_by_start_time = high_resolution_clock::now();
    // printf("Gathering offset\n");
    uint64_t** group_byte_offset = new uint64_t*[num_keys];
    uint64_t* distinct_bound = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
    uint64_t* group_idx = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
    uint64_t* d_num_bytes = gpuBufferManager->customCudaMalloc<uint64_t>(num_keys, 0, 0);

    for (uint64_t key = 0; key < num_keys; key++) {
        uint64_t* temp = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
        group_byte_offset[key] = gpuBufferManager->customCudaMalloc<uint64_t>(N + 1, 0, 0);
        hipMemset(group_byte_offset[key] + N, 0, sizeof(uint64_t));

        gather_and_modify<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(len[key], temp, materialized_temp, N, meta_num_keys);
        CHECK_ERROR();
        distinct_string<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(distinct_bound, temp, temp, materialized_temp, N);
        CHECK_ERROR();
        //cub scan
        d_temp_storage = nullptr;
        temp_storage_bytes = 0;
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp, group_byte_offset[key], N + 1);

        // Allocate temporary storage for exclusive prefix sum
        d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

        // Run exclusive prefix sum
        hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, temp, group_byte_offset[key], N + 1);
        CHECK_ERROR();

        hipMemcpy(d_num_bytes + key, group_byte_offset[key] + N, sizeof(uint64_t), hipMemcpyDeviceToDevice);
        CHECK_ERROR();
    }

    //copy num_bytes over
    hipMemcpy(num_bytes, d_num_bytes, num_keys * sizeof(uint64_t), hipMemcpyDeviceToHost);
    uint64_t** group_byte_offset_dev;
    hipMalloc((void**) &group_byte_offset_dev, num_keys * sizeof(uint64_t*));
    hipMemcpy(group_byte_offset_dev, group_byte_offset, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

    //cub scan
    d_temp_storage = nullptr;
    temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, distinct_bound, group_idx, N + 1);

    // Allocate temporary storage for exclusive prefix sum
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, distinct_bound, group_idx, N + 1);
    CHECK_ERROR();

    //gather the aggregates based on the row_sequence
    // printf("Gathering Aggregates\n");
    V** aggregate_keys_temp = new V*[num_aggregates];
    uint64_t** aggregate_star_temp = new uint64_t*[num_aggregates];
    sort_keys_type_string* group_by_rows = reinterpret_cast<sort_keys_type_string*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0));
    uint64_t* d_num_runs_out = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
    uint64_t* h_count = new uint64_t[1];

    for (int agg = 0; agg < num_aggregates; agg++) {
        // printf("Aggregating %d\n", agg);
        hipMemset(d_num_runs_out, 0, sizeof(uint64_t));
        if (agg_mode[agg] == 4 || agg_mode[agg] == 5) { //count_star or count(null) or sum(null)
            aggregate_star_temp[agg] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
            if (agg_mode[agg] == 4) {
                fill_n<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(aggregate_star_temp[agg], 1, N);
            } else if (agg_mode[agg] == 5) {
                hipMemset(aggregate_star_temp[agg], 0, N * sizeof(double));
            }

            modify<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(materialized_temp, N, meta_num_keys);
            CHECK_ERROR();

            //perform reduce_by_key
            uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
            hipMemset(agg_star_out, 0, N * sizeof(uint64_t));

            // printf("Reduce by key count_star\n");
            // Determine temporary device storage requirements
            d_temp_storage = nullptr;
            temp_storage_bytes = 0;
            CustomSumString custom_sum;
            hipcub::DeviceReduce::ReduceByKey(
                d_temp_storage, temp_storage_bytes,
                materialized_temp, group_by_rows, aggregate_star_temp[agg],
                agg_star_out, d_num_runs_out, custom_sum, N);

            CHECK_ERROR();

            // Allocate temporary storage
            d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

            // Run reduce-by-key
            hipcub::DeviceReduce::ReduceByKey(
                d_temp_storage, temp_storage_bytes,
                materialized_temp, group_by_rows, aggregate_star_temp[agg],
                agg_star_out, d_num_runs_out, custom_sum, N);

            CHECK_ERROR();

            hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
            count[0] = h_count[0];

            // printf("Count: %lu\n", count[0]);

            CHECK_ERROR();
            aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_star_out);
        } else {
            aggregate_keys_temp[agg] = gpuBufferManager->customCudaMalloc<V>(N, 0, 0);
            V* temp = reinterpret_cast<V*> (aggregate_keys[agg]);
            gather_and_modify<V, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(temp, aggregate_keys_temp[agg], materialized_temp, N, meta_num_keys);
            CHECK_ERROR();

            V* agg_out = gpuBufferManager->customCudaMalloc<V>(N, 0, 0);
            hipMemset(agg_out, 0, N * sizeof(V));

            CHECK_ERROR();
            if (agg_mode[agg] == 0) {
                // printf("Reduce by key sum\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomSumString custom_sum;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_out);
                // printf("Count: %lu\n", count[0]);
            } else if (agg_mode[agg] == 1) {
                //Currently typename V has to be a double
                // printf("Reduce by key avg\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomSumString custom_sum;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                aggregate_star_temp[agg] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
                fill_n<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(aggregate_star_temp[agg], 1, N);

                uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
                hipMemset(agg_star_out, 0, N * sizeof(uint64_t));
                hipMemset(d_num_runs_out, 0, sizeof(uint64_t));

                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_star_temp[agg],
                    agg_star_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_star_temp[agg],
                    agg_star_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                V* output = gpuBufferManager->customCudaMalloc<V>(count[0], 0, 0);
                divide<V, BLOCK_THREADS, ITEMS_PER_THREAD><<<(count[0] + tile_items - 1)/tile_items, BLOCK_THREADS>>>(agg_out, agg_star_out, output, count[0]);

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (output);
            } else if (agg_mode[agg] == 2) {
                // printf("Reduce by key max\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomMaxString custom_max;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_max, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_max, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_out);
            } else if (agg_mode[agg] == 3) {
                // printf("Reduce by key min\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomMinString custom_min;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_min, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0));

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_min, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_out);
            }
        }
    }
    hipDeviceSynchronize();
    CHECK_ERROR();

    auto group_by_end_time = high_resolution_clock::now();
    auto group_by_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(group_by_end_time - group_by_start_time).count();
    std::cout << "STRING GROUP BY V1: Group By took " << group_by_time_ms << " ms" << std::endl;

    auto post_processing_start_time = high_resolution_clock::now();
    uint64_t** offset_dev_result;
    hipMalloc((void**) &offset_dev_result, num_keys * sizeof(uint64_t*));
    for (uint64_t i = 0; i < num_keys; i++) {
        offset[i] = gpuBufferManager->customCudaMalloc<uint64_t>(count[0], 0, 0);
    }
    hipMemcpy(offset_dev_result, offset, num_keys * sizeof(uint8_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    compact_string_offset<BLOCK_THREADS, ITEMS_PER_THREAD><<<((N + 1) + tile_items - 1)/tile_items, BLOCK_THREADS>>>(
            group_idx, group_byte_offset_dev, offset_dev_result, N + 1, num_keys);

    CHECK_ERROR();

    uint8_t** keys_dev_result;
    hipMalloc((void**) &keys_dev_result, num_keys * sizeof(uint8_t*));
    for (uint64_t i = 0; i < num_keys; i++) {
        uint64_t* temp_num_bytes = new uint64_t[1];
        hipMemcpy(temp_num_bytes, offset[i] + count[0], sizeof(uint64_t), hipMemcpyDeviceToHost);
        keys[i] = gpuBufferManager->customCudaMalloc<uint8_t>(temp_num_bytes[0], 0, 0);
    }
    hipMemcpy(keys_dev_result, keys, num_keys * sizeof(uint8_t*), hipMemcpyHostToDevice);
    CHECK_ERROR();

    rows_to_columns_string<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(
            group_idx, group_by_rows, keys_dev_result, group_byte_offset_dev, key_length, N, num_keys);

    hipDeviceSynchronize();
    CHECK_ERROR();
    auto post_processing_end_time = high_resolution_clock::now();
    auto post_processing_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(post_processing_end_time - post_processing_start_time).count();
    std::cout << "STRING GROUP BY V1: Post Processing took " << post_processing_time_ms << " ms" << std::endl;

    // testprint<uint64_t><<<1, 1>>>(group_idx, N);
    // testprint<double><<<1, 1>>>(reinterpret_cast<double*> (aggregate_keys[0]), N);
    // testprint<uint64_t><<<1, 1>>>(offset[1], N);
    // CHECK_ERROR();

    hipDeviceSynchronize();
    printf("Count: %lu\n", count[0]);
    throw std::runtime_error("Grouped String Aggregate V1 implementation stop");
}

__global__ void add_offset(uint64_t* a, uint64_t* b, uint64_t offset, uint64_t N) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        a[idx] = b[idx] + offset;
    }
}

void combineStrings(uint8_t* a, uint8_t* b, uint8_t* c, 
        uint64_t* offset_a, uint64_t* offset_b, uint64_t* offset_c, 
        uint64_t num_bytes_a, uint64_t num_bytes_b, uint64_t N_a, uint64_t N_b) {
    CHECK_ERROR();
    if (N_a == 0 || N_b == 0) {
        printf("N is 0\n");
        return;
    }
    hipMemcpy(c, a, num_bytes_a * sizeof(uint8_t), hipMemcpyDeviceToDevice);
    hipMemcpy(c + num_bytes_a, b, num_bytes_b * sizeof(uint8_t), hipMemcpyDeviceToDevice);

    hipMemcpy(offset_c, offset_a, N_a * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    add_offset<<<((N_b + 1) + BLOCK_THREADS - 1)/(BLOCK_THREADS), BLOCK_THREADS>>>(offset_c + N_a, offset_b, num_bytes_a, N_b + 1);
    CHECK_ERROR();
    hipDeviceSynchronize();
}

template
void groupedStringAggregate<double>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

template
void groupedStringAggregate<uint64_t>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

}