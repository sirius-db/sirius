#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_physical_hash_join.hpp"
#include "gpu_buffer_manager.hpp"

namespace duckdb {

__device__ uint64_t hash64_multikey(uint64_t key1, uint64_t key2) {
    uint64_t h = key1 * 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    h ^= key2 * 0xc6a4a7935bd1e995ull;
    h *= 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    return h;
}

__device__ uint64_t hash32_multikey(int32_t key1, int32_t key2) {
    uint64_t k1 = (uint64_t)key1;
    uint64_t k2 = (uint64_t)key2;
    uint64_t h = k1 * 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    h ^= k2 * 0xc6a4a7935bd1e995ull;
    h *= 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    return h;
}


template <typename T, int B, int I>
__global__ void probe_multikey_count_t(T **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread,
        unsigned long long* total_count, uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right) {
    typedef hipcub::BlockScan<int, B> BlockScanInt;

    __shared__ union TempStorage
    {
        typename BlockScanInt::TempStorage scan;
    } temp_storage;

    int items_count[I];

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    int t_count = 0; // Number of items selected per thread
    int c_t_count = 0; //Prefix sum of t_count

    __shared__ uint64_t block_off;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        items_count[ITEM] = 0;
    }

    int n_ht_column;
    if (is_right) n_ht_column = num_keys + 2;
    else n_ht_column = num_keys + 1;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {

            uint64_t slot;
            if (equal_keys == 1) {
                T k0 = keys[0][tile_offset + threadIdx.x + ITEM * B];
                if (sizeof(T) == 4) {
                    slot = (int32_t)(k0) % ht_len;
                } else {
                    slot = ((uint64_t)k0) % ht_len;
                }
            } else if (equal_keys == 2) {
                T k0 = keys[0][tile_offset + threadIdx.x + ITEM * B];
                T k1 = keys[1][tile_offset + threadIdx.x + ITEM * B];
                if (sizeof(T) == 4) {
                    slot = hash32_multikey((int32_t)k0, (int32_t)k1) % ht_len;
                } else {
                    slot = hash64_multikey((uint64_t)k0, (uint64_t)k1) % ht_len;
                }
            } else {
                cudaAssert(0);
            }

            // if (keys[0][tile_offset + threadIdx.x + ITEM * B] == 4701966275692616012 || keys[0][tile_offset + threadIdx.x + ITEM * B] == 4701966275692616011) {
            //     printf("key found %ld\n", tile_offset + threadIdx.x + ITEM * B);
            // }

            // if (tile_offset + threadIdx.x + ITEM * B == 69997) {
            //     printf("key %ld\n", keys[0][tile_offset + threadIdx.x + ITEM * B]);
            // }

            while (ht[slot * n_ht_column] != 0xFFFFFFFFFFFFFFFF) {
                bool local_found = 1;
                // printf("key1: %lu key2: %lu ht1: %lu ht2: %lu\n", keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B], ht[slot * n_ht_column], ht[slot * n_ht_column + 1]);
                for (int n = 0; n < num_keys; n++) {
                    T item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0 && item != ht[slot * n_ht_column + n]) {
                            local_found = 0;  
                    } else if (condition_mode[n] == 1 && item == ht[slot * n_ht_column + n]) {
                            local_found = 0;
                            // printf("key1: %lu key2: %lu ht1: %lu ht2: %lu\n", keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B], ht[slot * n_ht_column], ht[slot * n_ht_column + 1]);
                            // break;
                    }
                }
                if (local_found) {
                    items_count[ITEM]++;
                }
                slot = (slot + 100007) % ht_len;
            }
            t_count += items_count[ITEM];
        }
    }

    //Barrier
    __syncthreads();
    BlockScanInt(temp_storage.scan).ExclusiveSum(t_count, c_t_count); //doing a prefix sum of all the previous threads in the block and store it to c_t_count

    if(threadIdx.x == blockDim.x - 1) { //if the last thread in the block, add the prefix sum of all the prev threads + sum of my threads to global variable total

        block_off = atomicAdd(total_count, (unsigned long long) t_count+c_t_count); //the previous value of total is gonna be assigned to block_off

    } //block_off does not need to be global (it's just need to be shared), because it will get the previous value from total which is global

    __syncthreads();

    if (blockIdx.x * tile_size + threadIdx.x < N) {
        offset_each_thread[blockIdx.x * B + threadIdx.x] = block_off + c_t_count;
    }
}

template <typename T, int B, int I>
__global__ void probe_multikey_t(
    T **keys,
    unsigned long long* ht,
    uint64_t ht_len,
    uint64_t *offset_each_thread,
    uint64_t *row_ids_left,
    uint64_t *row_ids_right,
    uint64_t N,
    int* condition_mode,
    int num_keys,
    int equal_keys,
    bool is_right
) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;
    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;
    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    int n_ht_column;
    if (is_right) n_ht_column = num_keys + 2;
    else n_ht_column = num_keys + 1;

    uint64_t output_offset = 0;
    if (blockIdx.x * tile_size + threadIdx.x < N) {
        output_offset = offset_each_thread[blockIdx.x * B + threadIdx.x];
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t slot;
            if (equal_keys == 1) {
                T k0 = keys[0][tile_offset + threadIdx.x + ITEM * B];
                if (sizeof(T) == 4) {
                    slot = (int32_t)(k0) % ht_len;
                } else {
                    slot = ((uint64_t)k0) % ht_len;
                }
            } else if (equal_keys == 2) {
                T k0 = keys[0][tile_offset + threadIdx.x + ITEM * B];
                T k1 = keys[1][tile_offset + threadIdx.x + ITEM * B];
                if (sizeof(T) == 4) {
                    slot = hash32_multikey((int32_t)k0, (int32_t)k1) % ht_len;
                } else {
                    slot = hash64_multikey((uint64_t)k0, (uint64_t)k1) % ht_len;
                }
            } else {
                cudaAssert(0);
            }
            while (ht[slot * n_ht_column] != 0xFFFFFFFFFFFFFFFFULL) {
                bool local_found = 1;
                for (int n = 0; n < num_keys; n++) {
                    T item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0) {
                        if ((uint64_t)item != ht[slot * n_ht_column + n]) {
                            local_found = 0;
                            // break;
                        }
                    } else if (condition_mode[n] == 1) {
                        if ((uint64_t)item == ht[slot * n_ht_column + n]) {
                            local_found = 0;
                            // break;
                        }
                    }
                }
                if (local_found) {
                    row_ids_right[output_offset] = ht[slot * n_ht_column + num_keys];
                    row_ids_left[output_offset]  = tile_offset + threadIdx.x + ITEM * B;
                    if (is_right) ht[slot * n_ht_column + num_keys + 1] = tile_offset + threadIdx.x + ITEM * B;
                    
                    output_offset++;
                }
                slot = (slot + 100007) % ht_len;
            }
        }
    }
}

template <typename T, int B, int I>
__global__ void build_multikey_t(
    T **keys,
    unsigned long long* ht,
    uint64_t ht_len,
    uint64_t N,
    int num_keys,
    int equal_keys,
    bool is_right
) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;
    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;
    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    int n_ht_column;
    if (is_right) n_ht_column = num_keys + 2;
    else  n_ht_column = num_keys + 1;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            uint64_t slot;
            if (equal_keys == 1) {
                T k0 = keys[0][tile_offset + threadIdx.x + ITEM * B];
                if (sizeof(T) == 4) {
                    slot = (int32_t)(k0) % ht_len;
                } else {
                    slot = ((uint64_t)k0) % ht_len;
                }
            } else if (equal_keys == 2) {
                T k0 = keys[0][tile_offset + threadIdx.x + ITEM * B];
                T k1 = keys[1][tile_offset + threadIdx.x + ITEM * B];
                if (sizeof(T) == 4) {
                    slot = hash32_multikey((int32_t)k0, (int32_t)k1) % ht_len;
                } else {
                    slot = hash64_multikey((uint64_t)k0, (uint64_t)k1) % ht_len;
                }
            } else {
                cudaAssert(0);
            }

            T item = keys[0][tile_offset + threadIdx.x + ITEM * B];
            while (atomicCAS(&ht[slot * n_ht_column], 0xFFFFFFFFFFFFFFFFULL,
                            (unsigned long long)( (uint64_t)item ))
                != 0xFFFFFFFFFFFFFFFFULL)
            {
            slot = (slot + 100007ULL) % ht_len;
            }
            for (int n = 1; n < num_keys; n++) {
                ht[slot * n_ht_column + n] = keys[n][tile_offset + threadIdx.x + ITEM * B];
            }
            ht[slot * n_ht_column + num_keys] = tile_offset + threadIdx.x + ITEM * B;
        }
    }
}

template __global__ void build_multikey_t<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(
    uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N,
    int num_keys, int equal_keys, bool is_right
);

template __global__ void build_multikey_t<int32_t, BLOCK_THREADS, ITEMS_PER_THREAD>(
    int32_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N,
    int num_keys, int equal_keys, bool is_right
);

template __global__ void probe_multikey_count_t<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(
    uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread,
    unsigned long long* total_count, uint64_t N, int* condition_mode,
    int num_keys, int equal_keys, bool is_right
);

template __global__ void probe_multikey_count_t<int32_t, BLOCK_THREADS, ITEMS_PER_THREAD>(
    int32_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread,
    unsigned long long* total_count, uint64_t N, int* condition_mode,
    int num_keys, int equal_keys, bool is_right
);

template __global__ void probe_multikey_t<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(
    uint64_t **keys, unsigned long long* ht, uint64_t ht_len,
    uint64_t *offset_each_thread, uint64_t *row_ids_left, uint64_t *row_ids_right,
    uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right
);

template __global__ void probe_multikey_t<int32_t, BLOCK_THREADS, ITEMS_PER_THREAD>(
    int32_t **keys, unsigned long long* ht, uint64_t ht_len,
    uint64_t *offset_each_thread, uint64_t *row_ids_left, uint64_t *row_ids_right,
    uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right
);

void buildHashTableTempl(
    uint8_t **keys,
    unsigned long long* ht,
    uint64_t ht_len,
    uint64_t N,
    int* condition_mode,
    int num_keys,
    bool is_right,
    bool is_32_bit  // added int 32 parameter
) {
    CHECK_ERROR();
    if (N == 0) {
        printf("N is 0\n");
        return;
    }
    printf("Launching Build Kernel\n");
    SETUP_TIMING();
    START_TIMER();
    printf("N: %lu ht_len: %ld\n", N, ht_len);

    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    int* condition_mode_dev = gpuBufferManager->customCudaMalloc<int>(num_keys, 0, 0);
    hipMemcpy(condition_mode_dev, condition_mode, num_keys * sizeof(int), hipMemcpyHostToDevice);

    int equal_keys = 0;
    for (int idx = 0; idx < num_keys; idx++) {
        if (condition_mode[idx] == 0) equal_keys++;
    }

    if (is_right) hipMemset(ht, 0xFF, ht_len * (num_keys + 2) * sizeof(unsigned long long));
    else hipMemset(ht, 0xFF, ht_len * (num_keys + 1) * sizeof(unsigned long long));
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;

    if (is_32_bit) {
        int32_t** keys_data = new int32_t*[num_keys];
        for (int i = 0; i < num_keys; i++) {
            keys_data[i] = reinterpret_cast<int32_t*>(keys[i]);
        }
        int32_t** keys_dev;
        hipMalloc(&keys_dev, num_keys * sizeof(int32_t*));
        hipMemcpy(keys_dev, keys_data, num_keys * sizeof(int32_t*), hipMemcpyHostToDevice);

        build_multikey_t<int32_t, BLOCK_THREADS, ITEMS_PER_THREAD>
            <<< (N + tile_items - 1)/tile_items, BLOCK_THREADS >>>(
                keys_dev, ht, ht_len, N, num_keys, equal_keys, is_right
            );
        hipDeviceSynchronize();
    } else {
        uint64_t** keys_data = new uint64_t*[num_keys];
        for (int i = 0; i < num_keys; i++) {
            keys_data[i] = reinterpret_cast<uint64_t*>(keys[i]);
        }
        uint64_t** keys_dev;
        hipMalloc(&keys_dev, num_keys * sizeof(uint64_t*));
        hipMemcpy(keys_dev, keys_data, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

        build_multikey_t<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>
            <<< (N + tile_items - 1)/tile_items, BLOCK_THREADS >>>(
                keys_dev, ht, ht_len, N, num_keys, equal_keys, is_right
            );
        hipDeviceSynchronize();
    }

    STOP_TIMER();
}

void probeHashTableTempl(
    uint8_t **keys,
    unsigned long long* ht,
    uint64_t ht_len,
    uint64_t*& row_ids_left,
    uint64_t*& row_ids_right,
    uint64_t*& count,
    uint64_t N,
    int* condition_mode,
    int num_keys,
    bool is_right,
    bool is_32_bit  
) {
    CHECK_ERROR();
    if (N == 0) {
        uint64_t* h_count = new uint64_t[1];
        h_count[0] = 0;
        count = h_count;
        printf("N is 0\n");
        return;
    }
    printf("Launching Probe Kernel\n");
    SETUP_TIMING();
    START_TIMER();
    printf("N: %lu\n", N);

    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;

    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    hipMemset(count, 0, sizeof(uint64_t));

    uint64_t* offset_each_thread = gpuBufferManager->customCudaMalloc<uint64_t>(((N + tile_items - 1)/tile_items) * BLOCK_THREADS, 0, 0);

    int* condition_mode_dev = gpuBufferManager->customCudaMalloc<int>(num_keys, 0, 0);
    hipMemcpy(condition_mode_dev, condition_mode, num_keys * sizeof(int), hipMemcpyHostToDevice);

    int equal_keys = 0;
    for (int i = 0; i < num_keys; i++) {
        if (condition_mode[i] == 0) equal_keys++;
    }

    if (is_32_bit) {
        int32_t** keys_data = new int32_t*[num_keys];
        for (int i = 0; i < num_keys; i++) {
            keys_data[i] = reinterpret_cast<int32_t*>(keys[i]);
        }
        int32_t** keys_dev;
        hipMalloc(&keys_dev, num_keys * sizeof(int32_t*));
        hipMemcpy(keys_dev, keys_data, num_keys * sizeof(int32_t*), hipMemcpyHostToDevice);

        probe_multikey_count_t<int32_t, BLOCK_THREADS, ITEMS_PER_THREAD>
            <<< (N + tile_items - 1)/tile_items, BLOCK_THREADS >>>(
                keys_dev, ht, ht_len,
                offset_each_thread, (unsigned long long*)count,
                N, condition_mode_dev, num_keys, equal_keys, is_right
            );
        CHECK_ERROR();
        hipDeviceSynchronize();

        uint64_t* h_count = new uint64_t[1];
        hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
        assert(h_count[0] > 0);
        printf("Count: %lu\n", h_count[0]);
        row_ids_left  = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
        row_ids_right = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
        probe_multikey_t<int32_t, BLOCK_THREADS, ITEMS_PER_THREAD>
            <<< (N + tile_items - 1)/tile_items, BLOCK_THREADS >>>(
                keys_dev, ht, ht_len,
                offset_each_thread,
                row_ids_left, row_ids_right,
                N, condition_mode_dev, num_keys, equal_keys, is_right
            );
        CHECK_ERROR();
        hipDeviceSynchronize();
        count = h_count;
    } else {
        uint64_t** keys_data = new uint64_t*[num_keys];
        for (int i = 0; i < num_keys; i++) {
            keys_data[i] = reinterpret_cast<uint64_t*>(keys[i]);
        }
        uint64_t** keys_dev;
        hipMalloc(&keys_dev, num_keys * sizeof(uint64_t*));
        hipMemcpy(keys_dev, keys_data, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

        probe_multikey_count_t<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>
            <<< (N + tile_items - 1)/tile_items, BLOCK_THREADS >>>(
                keys_dev, ht, ht_len,
                offset_each_thread, (unsigned long long*)count,
                N, condition_mode_dev, num_keys, equal_keys, is_right
            );
        CHECK_ERROR();
        hipDeviceSynchronize();

        uint64_t* h_count = new uint64_t[1];
        hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
        assert(h_count[0] > 0);
        printf("Count: %lu\n", h_count[0]);

        row_ids_left  = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
        row_ids_right = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);

        probe_multikey_t<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>
            <<< (N + tile_items - 1)/tile_items, BLOCK_THREADS >>>(
                keys_dev, ht, ht_len,
                offset_each_thread,
                row_ids_left, row_ids_right,
                N, condition_mode_dev, num_keys, equal_keys, is_right
            );
        CHECK_ERROR();
        hipDeviceSynchronize();
        count = h_count;
    }

    STOP_TIMER();
}


void buildHashTable(uint8_t **keys,
                    unsigned long long* ht,
                    uint64_t ht_len,
                    uint64_t N,
                    int* condition_mode,
                    int num_keys,
                    bool is_right,
                    bool is_32_bit)
{
    buildHashTableTempl(keys, ht, ht_len, N, condition_mode, num_keys, is_right, is_32_bit);
}

void probeHashTable(uint8_t **keys,
                    unsigned long long* ht,
                    uint64_t ht_len,
                    uint64_t* &row_ids_left,
                    uint64_t* &row_ids_right,
                    uint64_t* &count,
                    uint64_t N,
                    int* condition_mode,
                    int num_keys,
                    bool is_right,
                    bool is_32_bit)
{
    probeHashTableTempl(keys, ht, ht_len,
                        row_ids_left, row_ids_right, count,
                        N, condition_mode,
                        num_keys, is_right,
                        is_32_bit);
}

} // namespace duckdb
