#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_physical_hash_join.hpp"
#include "gpu_buffer_manager.hpp"

namespace duckdb {

__device__ uint64_t hash64_multikey(uint64_t key1, uint64_t key2) {
    uint64_t h = key1 * 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    h ^= key2 * 0xc6a4a7935bd1e995ull;
    h *= 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    return h;
}

template <int B, int I>
__global__ void probe_multikey_count(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread, 
            unsigned long long* total_count, uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right) {

    typedef hipcub::BlockScan<int, B> BlockScanInt;

    __shared__ union TempStorage
    {
        typename BlockScanInt::TempStorage scan;
    } temp_storage;

    int items_count[I];

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    int t_count = 0; // Number of items selected per thread
    int c_t_count = 0; //Prefix sum of t_count
    __shared__ uint64_t block_off;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        items_count[ITEM] = 0;
    }

    int n_ht_column;
    if (is_right) n_ht_column = num_keys + 2;
    else n_ht_column = num_keys + 1;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            
            uint64_t slot;
            if (equal_keys == 1) slot = keys[0][tile_offset + threadIdx.x + ITEM * B] % ht_len;
            else if (equal_keys == 2) slot = hash64_multikey(keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B]) % ht_len;
            else cudaAssert(0);
            
            while (ht[slot * n_ht_column] != 0xFFFFFFFFFFFFFFFF) {
                bool local_found = 1;
                // printf("key1: %lu key2: %lu ht1: %lu ht2: %lu\n", keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B], ht[slot * n_ht_column], ht[slot * n_ht_column + 1]);
                for (int n = 0; n < num_keys; n++) {
                    uint64_t item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0 && ht[slot * n_ht_column + n] != item) {
                        local_found = 0;
                        // break;
                    } else if (condition_mode[n] == 1 && ht[slot * n_ht_column + n] == item) {
                        local_found = 0;
                        // printf("key1: %lu key2: %lu ht1: %lu ht2: %lu\n", keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B], ht[slot * n_ht_column], ht[slot * n_ht_column + 1]);
                        // break;
                    }
                }
                if (local_found) {
                    items_count[ITEM]++;
                }
                slot = (slot + 100007) % ht_len;
            }
            t_count += items_count[ITEM];
        }
    }

    //Barrier
    __syncthreads();

    BlockScanInt(temp_storage.scan).ExclusiveSum(t_count, c_t_count); //doing a prefix sum of all the previous threads in the block and store it to c_t_count
    if(threadIdx.x == blockDim.x - 1) { //if the last thread in the block, add the prefix sum of all the prev threads + sum of my threads to global variable total
        block_off = atomicAdd(total_count, (unsigned long long) t_count+c_t_count); //the previous value of total is gonna be assigned to block_off
    } //block_off does not need to be global (it's just need to be shared), because it will get the previous value from total which is global

    __syncthreads();

     if (blockIdx.x * tile_size + threadIdx.x < N) {
        offset_each_thread[blockIdx.x * B + threadIdx.x] = block_off + c_t_count;
    }

}

template <int B, int I>
__global__ void probe_multikey(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread, 
        uint64_t *row_ids_left, uint64_t *row_ids_right, uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    int n_ht_column;
    if (is_right) n_ht_column = num_keys + 2;
    else n_ht_column = num_keys + 1;

    uint64_t output_offset = 0;
    if (blockIdx.x * tile_size + threadIdx.x < N) {
        output_offset = offset_each_thread[blockIdx.x * B + threadIdx.x];
    }
    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            uint64_t slot;
            if (equal_keys == 1) slot = keys[0][tile_offset + threadIdx.x + ITEM * B] % ht_len;
            else if (equal_keys == 2) slot = hash64_multikey(keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B]) % ht_len;
            else cudaAssert(0);
            
            bool found = 0;
            while (ht[slot * n_ht_column] != 0xFFFFFFFFFFFFFFFF) {
                bool local_found = 1;
                for (int n = 0; n < num_keys; n++) {
                    uint64_t item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0 && ht[slot * n_ht_column + n] != item) {
                        local_found = 0;
                        // break;
                    } else if (condition_mode[n] == 1 && ht[slot * n_ht_column + n] == item) {
                        local_found = 0;
                        // break;
                    }
                }
                if (local_found) {
                    row_ids_right[output_offset] = ht[slot * n_ht_column + num_keys];
                    row_ids_left[output_offset] = tile_offset + threadIdx.x + ITEM * B;
                    if (is_right) ht[slot * n_ht_column + num_keys + 1] = tile_offset + threadIdx.x + ITEM * B;
                    output_offset++;
                }
                slot = (slot + 100007) % ht_len;
            }
        }
    }
}

template <int B, int I>
__global__ void build_multikey(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N, int num_keys, int equal_keys, bool is_right) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    int n_ht_column;
    if (is_right) n_ht_column = num_keys + 2;
    else  n_ht_column = num_keys + 1;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            uint64_t slot;
            if (equal_keys == 1) slot = keys[0][tile_offset + threadIdx.x + ITEM * B] % ht_len;
            else if (equal_keys == 2) slot = hash64_multikey(keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B]) % ht_len;
            else cudaAssert(0);
            
            uint64_t item = keys[0][tile_offset + threadIdx.x + ITEM * B];
            while(atomicCAS(&ht[slot * n_ht_column], 0xFFFFFFFFFFFFFFFF, (unsigned long long) item) != 0xFFFFFFFFFFFFFFFF) {                
                slot = (slot + 100007) % ht_len;
            }

            for (int n = 1; n < num_keys; n++) {
                ht[slot * n_ht_column + n] = keys[n][tile_offset + threadIdx.x + ITEM * B];
            }
            ht[slot * n_ht_column + num_keys] = tile_offset + threadIdx.x + (ITEM * B);
        }
    }
}

__global__ void print_hash_table(unsigned long long* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (uint64_t i = 0; i < 100; i++) {
            printf("%llu ", a[i]);
        }
        printf("\n");
    }
}

__global__ void print_key(uint64_t* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        int x = N;
        if (N > 100) x = 100;
        for (uint64_t i = 0; i < x; i++) {
            printf("%ld ", a[i]);
        }
        printf("\n");
    }
}

template
__global__ void build_multikey<BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N, int num_keys, int equal_keys, bool is_right);

template
__global__ void probe_multikey_count<BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread, 
            unsigned long long* total_count, uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right);

template
__global__ void probe_multikey<BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *offset_each_thread, 
        uint64_t *row_ids_left, uint64_t *row_ids_right, uint64_t N, int* condition_mode, int num_keys, int equal_keys, bool is_right);

void buildHashTable(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N, int* condition_mode, int num_keys, bool is_right) {
    CHECK_ERROR();
    if (N == 0) {
        printf("N is 0\n");
        return;
    }
    printf("Launching Build Kernel\n");
    printf("N: %lu\n", N);
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    //reinterpret cast the keys to uint64_t
    uint64_t** keys_data = new uint64_t*[num_keys];
    for (int idx = 0; idx < num_keys; idx++) {
        keys_data[idx] = reinterpret_cast<uint64_t*>(keys[idx]);
    }

    uint64_t** keys_dev;
    hipMalloc((void**) &keys_dev, num_keys * sizeof(uint64_t*));
    hipMemcpy(keys_dev, keys_data, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

    int equal_keys = 0;
    for (int idx = 0; idx < num_keys; idx++) {
        if (condition_mode[idx] == 0) equal_keys++;
    }


    if (is_right) hipMemset(ht, 0xFF, ht_len * (num_keys + 2) * sizeof(unsigned long long));
    else hipMemset(ht, 0xFF, ht_len * (num_keys + 1) * sizeof(unsigned long long));
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;

    // for (int idx = 0; idx < num_keys; idx++) {
    //     print_key<<<1, 1>>>(keys[idx], N);
    // }
    
    build_multikey<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, ht, ht_len, N, num_keys, equal_keys, is_right);
    // print_hash_table<<<1, 1>>>(ht, ht_len);
    CHECK_ERROR();
    hipDeviceSynchronize();
}

void probeHashTable(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t* &row_ids_left, uint64_t* &row_ids_right, uint64_t* &count, uint64_t N, int* condition_mode, int num_keys, bool is_right) {
    CHECK_ERROR();
    if (N == 0) {
        uint64_t* h_count = new uint64_t[1];
        h_count[0] = 0;
        count = h_count;
        printf("N is 0\n");
        return;
    }
    printf("Launching Probe Kernel\n");
    printf("N: %lu\n", N);
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    hipMemset(count, 0, sizeof(uint64_t));
    uint64_t* offset_each_thread = gpuBufferManager->customCudaMalloc<uint64_t>(((N + tile_items - 1)/tile_items) * BLOCK_THREADS, 0, 0).data_;

    //reinterpret cast the keys to uint64_t
    uint64_t** keys_data = new uint64_t*[num_keys];
    for (int idx = 0; idx < num_keys; idx++) {
        keys_data[idx] = reinterpret_cast<uint64_t*>(keys[idx]);
    }

    uint64_t** keys_dev;
    hipMalloc((void**) &keys_dev, num_keys * sizeof(uint64_t*));
    hipMemcpy(keys_dev, keys_data, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

    int equal_keys = 0;
    for (int idx = 0; idx < num_keys; idx++) {
        if (condition_mode[idx] == 0) equal_keys++;
    }

    int* condition_mode_dev = gpuBufferManager->customCudaMalloc<int>(num_keys, 0, 0).data_;
    hipMemcpy(condition_mode_dev, condition_mode, num_keys * sizeof(int), hipMemcpyHostToDevice);
    
    CHECK_ERROR();
    probe_multikey_count<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, ht, ht_len, 
            offset_each_thread, (unsigned long long*) count, N, condition_mode_dev, num_keys, equal_keys, is_right);
    CHECK_ERROR();
    hipDeviceSynchronize();

    uint64_t* h_count = new uint64_t[1];
    hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
    assert(h_count[0] > 0);
    printf("Count: %lu\n", h_count[0]);
    row_ids_left = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0).data_;
    row_ids_right = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0).data_;
    probe_multikey<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, ht, ht_len, 
            offset_each_thread, row_ids_left, row_ids_right, N, condition_mode_dev, num_keys, equal_keys, is_right);
    CHECK_ERROR();
    hipDeviceSynchronize();
    count = h_count;
}


} // namespace duckdb