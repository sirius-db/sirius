#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_physical_grouped_aggregate.hpp"
#include "gpu_buffer_manager.hpp"

#include <chrono>
#include <stdexcept>

namespace duckdb {

using std::chrono::high_resolution_clock;
using std::chrono::duration;

constexpr bool V3_LOG_MODE = false;
__device__ int d_comparator_keys_compared_v3 = 0;
__device__ int d_comparator_num_comparsions_v3 = 0;

struct sort_keys_type_string_v3 {
  uint64_t row_id;
  uint64_t row_signature;

  __host__ __device__ sort_keys_type_string_v3() {}
  __host__ __device__ sort_keys_type_string_v3(uint64_t _row_id, uint64_t _row_signature) : row_id(_row_id), row_signature(_row_signature) {}
};

struct CustomLessStringV3 {
    uint8_t** all_keys;
    uint64_t** offsets;
    uint64_t num_keys;

    __host__ __device__ CustomLessStringV3() {}
    __host__ __device__ CustomLessStringV3(uint8_t** _all_keys, uint64_t** _offsets, uint64_t _num_keys) : all_keys(_all_keys), offsets(_offsets),  num_keys(_num_keys) {}

    __device__ bool operator()(const sort_keys_type_string_v3& lhs, const sort_keys_type_string_v3& rhs) {
        if constexpr(V3_LOG_MODE) {
            atomicAdd(&d_comparator_num_comparsions_v3, (int) 1);
        }

        // First compare the signatures
        if (lhs.row_signature != rhs.row_signature) {
            if constexpr(V3_LOG_MODE) {
                atomicAdd(&d_comparator_keys_compared_v3, (int) 1);
            }
            return lhs.row_signature < rhs.row_signature;
        }

        // If the signature is the same then compare the invidiual lengths
        uint64_t values_compared = 1; 
        uint64_t left_val; uint64_t right_val;
        for(uint64_t i = 0; i < num_keys; i++) {
            uint64_t* curr_column_offsets = offsets[i];
            left_val = curr_column_offsets[lhs.row_id + 1] - curr_column_offsets[lhs.row_id];
            right_val = curr_column_offsets[rhs.row_id + 1] - curr_column_offsets[rhs.row_id];
            if(left_val != right_val) {
                if constexpr(V3_LOG_MODE) {
                    atomicAdd(&d_comparator_keys_compared_v3, (int) (values_compared + i));
                }
                return left_val < right_val;
            }
        }
        values_compared += num_keys;

        // If the lengths are the same then compare the individiual chars
        uint8_t curr_left_val; uint8_t curr_right_val;
        for(uint64_t i = 0; i < num_keys; i++) {
            // Get the offset details for this key
            uint64_t* curr_column_offsets = offsets[i];
            uint64_t left_read_offset = curr_column_offsets[lhs.row_id];
            uint64_t right_read_offset = curr_column_offsets[rhs.row_id];
            const uint64_t curr_length = curr_column_offsets[lhs.row_id + 1] - left_read_offset;

            // Determine the chars to compare
            uint8_t* curr_column_keys = all_keys[i];
            uint8_t* left_read_chars = curr_column_keys + left_read_offset;
            uint8_t* right_read_chars = curr_column_keys + right_read_offset;

            #pragma unroll
            for(uint64_t j = 0; j < curr_length; j++) {
                curr_left_val = left_read_chars[j]; curr_right_val = right_read_chars[j];
                if(curr_left_val != curr_right_val) {
                    if constexpr(V3_LOG_MODE) {
                        atomicAdd(&d_comparator_keys_compared_v3, (int) (values_compared + j));
                    }
                    return curr_left_val < curr_right_val;
                }
            }

            if constexpr(V3_LOG_MODE) {
                values_compared += curr_length;
            }
        }

        if constexpr(V3_LOG_MODE) {
            atomicAdd(&d_comparator_keys_compared_v3, (int) values_compared);
        }
        return true;
    }
}; 


__global__ void fill_preprocess_buffer(uint8_t** keys, uint64_t** column_length_offsets, sort_keys_type_string_v3* row_records, const uint64_t num_rows, const uint64_t num_keys) {
    const uint64_t tile_size = gridDim.x * blockDim.x;
    const uint64_t start_idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Create the record for the current row in a tile based manner
    uint64_t curr_value;
    for(uint64_t i = start_idx; i < num_rows; i += tile_size) {
        // Get the signature for this row
        uint64_t signature = 0;
        uint64_t curr_power = 1;
        #pragma unroll
        for(uint64_t j = 0; j < num_keys; j++) {
            // Get the chars for this row in this column
            uint64_t* curr_column_offsets = column_length_offsets[j];
            uint64_t curr_row_start = curr_column_offsets[i];
            uint64_t curr_record_length = curr_column_offsets[i + 1] - curr_row_start;
            uint8_t* column_hash_chars = keys[j] + curr_row_start;

            // Update the signature using this record
            #pragma unroll
            for(uint64_t k = 0; k < curr_record_length; k++) {
                curr_value = static_cast<uint64_t>(column_hash_chars[k]);
                signature = (signature + curr_value * curr_power) % HASH_MOD_VALUE;
                curr_power = (curr_power * HASH_POWER) % HASH_MOD_VALUE;
            }
        }

        row_records[i] = sort_keys_type_string_v3(i, signature);
    }
}

__global__ void print_sort_metadata_v3() {
    float average_compare_values = (1.0 * d_comparator_keys_compared_v3)/d_comparator_num_comparsions_v3;
    printf("STRING GROUP BY V3: Performed %d row comparsions checking an average of %f values\n", d_comparator_num_comparsions_v3, average_compare_values);
}

template <typename V>
void groupedStringAggregateV3(uint8_t** keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode) {
    CHECK_ERROR();
    if (N == 0) {
        count[0] = 0;
        printf("N is 0\n");
        return;
    }

    printf("Launching String Grouped Aggregate Kernel V3\n");
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    // Create the temporary buffer
    uint64_t total_preprocessing_bytes = 2 * N * sizeof(uint64_t);
    auto preprocess_start_time = high_resolution_clock::now();

    sort_keys_type_string_v3* d_row_records = reinterpret_cast<sort_keys_type_string_v3*>(gpuBufferManager->customCudaMalloc<key_and_signature>(N, 0, 0));
    uint64_t items_per_block = BLOCK_THREADS * ITEMS_PER_THREAD;
    uint64_t num_blocks = (N + items_per_block - 1)/items_per_block;
    fill_preprocess_buffer<<<num_blocks, BLOCK_THREADS>>>(keys, offset, d_row_records, N, num_keys);

    hipDeviceSynchronize();
    CHECK_ERROR();
    auto preprocess_end_time = high_resolution_clock::now();
    auto preprocess_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(preprocess_end_time - preprocess_start_time).count();
    std::cout << "STRING GROUP BY V3: Preprocessing requires " << total_preprocessing_bytes << " bytes" << std::endl;
    std::cout << "STRING GROUP BY V3: Preprocessing took " << preprocess_time_ms << " ms" << std::endl;

    // Perform the sort
    auto sort_start_time = high_resolution_clock::now();

    CustomLessStringV3 custom_less_comparator(keys, offset, num_keys);
    void* sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;
    hipcub::DeviceMergeSort::SortKeys(
        sort_temp_storage,
        sort_temp_storage_bytes,
        d_row_records,
        N,
        custom_less_comparator);

    CHECK_ERROR();

    // Allocate temporary storage
    sort_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(sort_temp_storage_bytes, 0, 0));

    // Run sorting operation
    hipcub::DeviceMergeSort::SortKeys(
        sort_temp_storage,
        sort_temp_storage_bytes,
        d_row_records,
        N,
        custom_less_comparator);

    hipDeviceSynchronize();
    CHECK_ERROR();
    auto sort_end_time = high_resolution_clock::now();
    auto sort_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(sort_end_time - sort_start_time).count();
    std::cout << "STRING GROUP BY V3: Sorting took " << sort_time_ms << " ms" << std::endl;
    print_sort_metadata_v3<<<1, 1>>>();

    auto group_by_start_time = high_resolution_clock::now();
    hipDeviceSynchronize();
    CHECK_ERROR();

    auto group_by_end_time = high_resolution_clock::now();
    auto group_by_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(group_by_end_time - group_by_start_time).count();
    std::cout << "STRING GROUP BY V3: Group By took " << group_by_time_ms << " ms" << std::endl;

    auto post_processing_start_time = high_resolution_clock::now();
    hipDeviceSynchronize();
    CHECK_ERROR();

    auto post_processing_end_time = high_resolution_clock::now();
    auto post_processing_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(post_processing_end_time - post_processing_start_time).count();
    std::cout << "STRING GROUP BY V3: Post Processing took " << post_processing_time_ms << " ms" << std::endl;

    throw std::runtime_error("Grouped String Aggregate V3 implementation incomplete");
}

template
void groupedStringAggregateV3<double>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

template
void groupedStringAggregateV3<uint64_t>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

}