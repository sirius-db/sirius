#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_physical_grouped_aggregate.hpp"
#include "gpu_buffer_manager.hpp"

#include <chrono>
#include <stdexcept>
#include <hipcub/hipcub.hpp>
#include <assert.h>

namespace duckdb
{

    using std::chrono::duration;
    using std::chrono::high_resolution_clock;

    struct string_groupby_metadata_v3
    {
        uint8_t **all_keys;
        uint64_t **offsets;
        uint64_t num_keys;

        __host__ __device__ string_groupby_metadata_v3() {}
        __host__ __device__ string_groupby_metadata_v3(uint8_t **_all_keys, uint64_t **_offsets, uint64_t _num_keys) : all_keys(_all_keys), offsets(_offsets), num_keys(_num_keys) {}
    };

    struct sort_keys_type_string_v3
    {
        string_groupby_metadata_v3 *group_by_metadata;
        uint64_t row_id;
        uint64_t row_signature;

        __host__ __device__ sort_keys_type_string_v3() {}
        __host__ __device__ sort_keys_type_string_v3(string_groupby_metadata_v3 *_metadata, uint64_t _row_id, uint64_t _row_signature) : group_by_metadata(_metadata), row_id(_row_id), row_signature(_row_signature) {}

        __device__ __forceinline__ bool operator==(const sort_keys_type_string_v3 &other) const
        {
            // First compare the signature
            if (row_signature != other.row_signature)
            {
                return false;
            }

            // Load the metadata into the local variables
            uint8_t **all_keys = this->group_by_metadata->all_keys;
            uint64_t **offsets = this->group_by_metadata->offsets;
            uint64_t num_keys = this->group_by_metadata->num_keys;

            // Then compare the lengths
            for (uint64_t i = 0; i < num_keys; i++)
            {
                uint64_t *curr_column_offsets = offsets[i];
                uint64_t left_length = curr_column_offsets[this->row_id + 1] - curr_column_offsets[this->row_id];
                uint64_t right_length = curr_column_offsets[other.row_id + 1] - curr_column_offsets[other.row_id];
                if (left_length != right_length)
                {
                    return false;
                }
            }

            // If the signature and lengths match then compare the actual values
            for (uint64_t i = 0; i < num_keys; i++)
            {
                // Read in the left and right offsets
                uint64_t *curr_column_offsets = offsets[i];
                const uint64_t left_read_offset = curr_column_offsets[this->row_id];
                const uint64_t right_read_offset = curr_column_offsets[other.row_id];
                const uint64_t curr_length = curr_column_offsets[this->row_id + 1] - left_read_offset;

                // Initialize left state
                uint64_t *curr_column_keys = reinterpret_cast<uint64_t *>(all_keys[i]);
                uint64_t left_int_idx = left_read_offset / BYTES_IN_INTEGER;
                uint64_t left_read_idx = left_read_offset % BYTES_IN_INTEGER;
                uint64_t curr_left_int = curr_column_keys[left_int_idx];

                // Initialize right state
                uint64_t right_int_idx = right_read_offset / BYTES_IN_INTEGER;
                uint64_t right_read_idx = right_read_offset % BYTES_IN_INTEGER;
                uint64_t curr_right_int = curr_column_keys[right_int_idx];

                uint64_t bytes_remaining = curr_length;
                while (bytes_remaining > 0)
                {
                    // Compare current batch of bytes
                    uint8_t batch_size = static_cast<uint8_t>(min(BYTES_IN_INTEGER - max(left_read_idx, right_read_idx), bytes_remaining));
                    uint8_t base_subtract_val = BYTES_IN_INTEGER - batch_size;
                    uint64_t keep_mask = (1ULL << (BITS_IN_BYTE * batch_size)) - 1;
                    uint8_t left_shift_val = base_subtract_val - left_read_idx;
                    uint64_t left_val = (curr_left_int >> (left_shift_val * BITS_IN_BYTE)) & keep_mask;
                    uint8_t right_shift_val = base_subtract_val - right_read_idx;
                    uint64_t right_val = (curr_right_int >> (right_shift_val * BITS_IN_BYTE)) & keep_mask;

                    // Now actually compare the values
                    if (left_val != right_val)
                    {
                        return false;
                    }

                    // Update trackers
                    bytes_remaining -= batch_size;

                    // Reload left integer if needed
                    left_read_idx += batch_size;
                    if (left_read_idx == BYTES_IN_INTEGER)
                    {
                        left_int_idx++;
                        curr_left_int = curr_column_keys[left_int_idx];
                        left_read_idx = 0;
                    }

                    // Reload right integer if needed
                    right_read_idx += batch_size;
                    if (right_read_idx == BYTES_IN_INTEGER)
                    {
                        right_int_idx++;
                        curr_right_int = curr_column_keys[right_int_idx];
                        right_read_idx = 0;
                    }
                }
            }

            return true;
        }

        __device__ __forceinline__ bool operator<(const sort_keys_type_string_v3 &other) const
        {
            // First compare the signature
            if (row_signature != other.row_signature)
            {
                return row_signature < other.row_signature;
            }

            // Load the metadata into the local variables
            uint8_t **all_keys = this->group_by_metadata->all_keys;
            uint64_t **offsets = this->group_by_metadata->offsets;
            uint64_t num_keys = this->group_by_metadata->num_keys;

            // Then compare the lengths
            for (uint64_t i = 0; i < num_keys; i++)
            {
                uint64_t *curr_column_offsets = offsets[i];
                uint64_t left_length = curr_column_offsets[this->row_id + 1] - curr_column_offsets[this->row_id];
                uint64_t right_length = curr_column_offsets[other.row_id + 1] - curr_column_offsets[other.row_id];
                if (left_length != right_length)
                {
                    return left_length < right_length;
                }
            }

            // If the signature and lengths match then compare the actual values
            for (uint64_t i = 0; i < num_keys; i++)
            {
                // Read in the left and right offsets
                uint64_t *curr_column_offsets = offsets[i];
                const uint64_t left_read_offset = curr_column_offsets[this->row_id];
                const uint64_t right_read_offset = curr_column_offsets[other.row_id];
                const uint64_t curr_length = curr_column_offsets[this->row_id + 1] - left_read_offset;

                // Initialize left state
                uint64_t *curr_column_keys = reinterpret_cast<uint64_t *>(all_keys[i]);
                uint64_t left_int_idx = left_read_offset / BYTES_IN_INTEGER;
                uint64_t left_read_idx = left_read_offset % BYTES_IN_INTEGER;
                uint64_t curr_left_int = curr_column_keys[left_int_idx];

                // Initialize right state
                uint64_t right_int_idx = right_read_offset / BYTES_IN_INTEGER;
                uint64_t right_read_idx = right_read_offset % BYTES_IN_INTEGER;
                uint64_t curr_right_int = curr_column_keys[right_int_idx];

                uint64_t bytes_remaining = curr_length;
                while (bytes_remaining > 0)
                {
                    // Compare current batch of bytes
                    uint8_t batch_size = static_cast<uint8_t>(min(BYTES_IN_INTEGER - max(left_read_idx, right_read_idx), bytes_remaining));
                    uint8_t base_subtract_val = BYTES_IN_INTEGER - batch_size;
                    uint64_t keep_mask = (1ULL << (BITS_IN_BYTE * batch_size)) - 1;
                    uint8_t left_shift_val = base_subtract_val - left_read_idx;
                    uint64_t left_val = (curr_left_int >> (left_shift_val * BITS_IN_BYTE)) & keep_mask;
                    uint8_t right_shift_val = base_subtract_val - right_read_idx;
                    uint64_t right_val = (curr_right_int >> (right_shift_val * BITS_IN_BYTE)) & keep_mask;

                    // Now actually compare the values
                    if (left_val != right_val)
                    {
                        return left_val < right_val;
                    }

                    // Update trackers
                    bytes_remaining -= batch_size;

                    // Reload left integer if needed
                    left_read_idx += batch_size;
                    if (left_read_idx == BYTES_IN_INTEGER)
                    {
                        left_int_idx++;
                        curr_left_int = curr_column_keys[left_int_idx];
                        left_read_idx = 0;
                    }

                    // Reload right integer if needed
                    right_read_idx += batch_size;
                    if (right_read_idx == BYTES_IN_INTEGER)
                    {
                        right_int_idx++;
                        curr_right_int = curr_column_keys[right_int_idx];
                        right_read_idx = 0;
                    }
                }
            }

            return true;
        }
    };

    struct CustomLessStringV3
    {
        __host__ __device__ CustomLessStringV3() {}

        __device__ __forceinline__ bool operator()(const sort_keys_type_string_v3 &lhs, const sort_keys_type_string_v3 &rhs)
        {
            return lhs < rhs;
        }
    };

    template <typename V>
    struct CustomCombineOperatorV3
    {

        V *aggregate_buffer_start;
        int *agg_mode;
        int num_aggregates;
        uint64_t N;

        __host__ CustomCombineOperatorV3(V *_buffer_start, int *_agg_mode, int _num_aggregates, uint64_t _N) : aggregate_buffer_start(_buffer_start), agg_mode(_agg_mode), num_aggregates(_num_aggregates), N(_N)
        {
        }

        __device__ __forceinline__ uint64_t operator()(const uint64_t &left, const uint64_t &right) const
        {
            // Get the lower and right from the left and the right
            uint64_t lower_idx = min(left, right);
            uint64_t upper_idx = max(left, right);
            if(lower_idx >= N || upper_idx >= N) {
                // We can't just skip because this currently occurs due to inactive threads calling this reduction operator
                // which means that the result of this is anyways going to be ignored.
                return upper_idx;
            }

            // Merge the upper records into the lower record
            V *lower_ptr = aggregate_buffer_start + lower_idx * (num_aggregates + 1);
            V *upper_ptr = aggregate_buffer_start + upper_idx * (num_aggregates + 1);
            lower_ptr[0] += upper_ptr[0];
            for (uint64_t i = 0; i < num_aggregates; i++)
            {
                if (agg_mode[i] == 2)
                {
                    lower_ptr[i + 1] = std::max(lower_ptr[i + 1], upper_ptr[i + 1]);
                }
                else if (agg_mode[i] == 3)
                {
                    lower_ptr[i + 1] = std::min(lower_ptr[i + 1], upper_ptr[i + 1]);
                }
                else
                {
                    lower_ptr[i + 1] += upper_ptr[i + 1];
                }
            }

            // Return the lower record
            return lower_idx;
        }
    };

    __global__ void create_metadata_record(string_groupby_metadata_v3 *group_by_metadata, uint8_t **keys, uint64_t **column_length_offsets, const uint64_t num_keys)
    {
        group_by_metadata->all_keys = keys;
        group_by_metadata->offsets = column_length_offsets;
        group_by_metadata->num_keys = num_keys;
    }

    __global__ void fill_preprocess_buffer(string_groupby_metadata_v3 *group_by_metadata, sort_keys_type_string_v3 *row_records, const uint64_t num_rows)
    {
        const uint64_t tile_size = gridDim.x * blockDim.x;
        const uint64_t start_idx = threadIdx.x + blockIdx.x * blockDim.x;

        // Create the record for the current row in a tile based manner
        uint64_t curr_value;
        uint64_t num_keys = group_by_metadata->num_keys;
        for (uint64_t i = start_idx; i < num_rows; i += tile_size)
        {
            // Get the signature for this row
            uint64_t signature = 0;
            uint64_t curr_power = 1;
#pragma unroll
            for (uint64_t j = 0; j < num_keys; j++)
            {
                // Get the chars for this row in this column
                uint64_t *curr_column_offsets = group_by_metadata->offsets[j];
                uint64_t curr_row_start = curr_column_offsets[i];
                uint64_t curr_record_length = curr_column_offsets[i + 1] - curr_row_start;
                uint8_t *column_hash_chars = group_by_metadata->all_keys[j] + curr_row_start;

// Update the signature using this record
#pragma unroll
                for (uint64_t k = 0; k < curr_record_length; k++)
                {
                    curr_value = static_cast<uint64_t>(column_hash_chars[k]);
                    signature = (signature + curr_value * curr_power) % HASH_MOD_VALUE;
                    curr_power = (curr_power * HASH_POWER) % HASH_MOD_VALUE;
                }
            }

            row_records[i] = sort_keys_type_string_v3(group_by_metadata, i, signature);
        }
    }

    template <typename V>
    __global__ void fill_aggregate_buffer(uint8_t** aggregate_input_keys, V *aggregate_write_buffer, uint64_t *aggregate_row_records,
                                          sort_keys_type_string_v3 *group_by_row_records, int *agg_mode, const uint64_t num_rows, const uint64_t num_aggregates)
    {

        const uint64_t tile_size = gridDim.x * blockDim.x;
        const uint64_t start_idx = threadIdx.x + blockIdx.x * blockDim.x;
        for (uint64_t i = start_idx; i < num_rows; i += tile_size)
        {
            uint64_t idx_row_id = group_by_row_records[i].row_id;

            // Copy over the aggregates into the buffers
            V *buffer_write_ptr = aggregate_write_buffer + i * (num_aggregates + 1);
            buffer_write_ptr[0] = static_cast<V>(1);
#pragma unroll
            for (uint64_t j = 0; j < num_aggregates; j++)
            {
                if (agg_mode[j] == 4)
                {
                    buffer_write_ptr[j + 1] = static_cast<V>(1);
                }
                else if (agg_mode[j] == 5)
                {
                    buffer_write_ptr[j + 1] = static_cast<V>(0);
                }
                else
                {
                    V* curr_aggregate_column = reinterpret_cast<V*>(aggregate_input_keys[j]);
                    buffer_write_ptr[j + 1] = curr_aggregate_column[idx_row_id];
                }
            }

            // Update the row record to contain the index to use to read these fields
            aggregate_row_records[i] = i;
        }
    }

    template <typename V>
    __global__ void perform_post_processing(uint64_t* result_aggregate_row_ids, uint64_t* group_row_ids, uint8_t** aggregate_input_keys, 
        V* aggregate_write_buffer, int* agg_mode, uint64_t num_groups, uint64_t num_aggregates) {
        
        const uint64_t curr_group_idx = threadIdx.x + blockIdx.x * blockDim.x;
        if(curr_group_idx < num_groups) {
            // Get the row id that this thread should copy over
            const uint64_t curr_row_id = result_aggregate_row_ids[curr_group_idx];
            group_row_ids[curr_group_idx] = curr_row_id;

            // Copy over the aggregates from the buffer back into the aggregate columns
            V* buffer_read_ptr = aggregate_write_buffer + curr_row_id * (num_aggregates + 1);
            V num_rows_in_group = buffer_read_ptr[0];
            
            #pragma unroll
            for (uint64_t i = 0; i < num_aggregates; i++) {
                V* curr_aggregate_column = reinterpret_cast<V*>(aggregate_input_keys[i]);
                curr_aggregate_column[curr_group_idx] = buffer_read_ptr[i + 1];
                if(agg_mode[i] == 1) {
                    curr_aggregate_column[curr_group_idx] /= num_rows_in_group;
                }
            }
        }
    }

    template <typename V>
    void groupedStringAggregateV3(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode)
    {
        CHECK_ERROR();
        if (N == 0)
        {
            count[0] = 0;
            printf("N is 0\n");
            return;
        }

        uint64_t max_value = std::numeric_limits<int>::max();
        if (N > max_value)
        {
            printf("String Group By currently only supported for at most %lu rows but got %lu rows\n", (uint64_t)max_value, (uint64_t)N);
            throw std::runtime_error("");
        }

        printf("Launching String Grouped Aggregate Kernel V3\n");
        GPUBufferManager *gpuBufferManager = &(GPUBufferManager::GetInstance());

        // First create the group by metadata
        uint64_t total_preprocessing_bytes = 2 * N * sizeof(uint64_t);
        auto preprocess_start_time = high_resolution_clock::now();
        string_groupby_metadata_v3 *d_group_by_metadata = reinterpret_cast<string_groupby_metadata_v3 *>(gpuBufferManager->customCudaMalloc<string_group_by_metadata_type>(1, 0, 0));
        create_metadata_record<<<1, 1>>>(d_group_by_metadata, keys, offset, num_keys);

        // Then create the row records using the metadata
        sort_keys_type_string_v3 *d_row_records = reinterpret_cast<sort_keys_type_string_v3 *>(gpuBufferManager->customCudaMalloc<pointer_and_two_values>(N, 0, 0));
        uint64_t items_per_block = BLOCK_THREADS * ITEMS_PER_THREAD;
        uint64_t num_blocks = (N + items_per_block - 1) / items_per_block;
        fill_preprocess_buffer<<<num_blocks, BLOCK_THREADS>>>(d_group_by_metadata, d_row_records, N);

        hipDeviceSynchronize();
        CHECK_ERROR();
        auto preprocess_end_time = high_resolution_clock::now();
        auto preprocess_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(preprocess_end_time - preprocess_start_time).count();
        std::cout << "STRING GROUP BY V3: Preprocessing required " << total_preprocessing_bytes << " bytes and took " << preprocess_time_ms << " ms" << std::endl;

        // Perform the sort
        auto sort_start_time = high_resolution_clock::now();

        CustomLessStringV3 custom_less_comparator;
        void *sort_temp_storage = nullptr;
        size_t sort_temp_storage_bytes = 0;
        hipcub::DeviceMergeSort::SortKeys(
            sort_temp_storage,
            sort_temp_storage_bytes,
            d_row_records,
            N,
            custom_less_comparator);

        hipDeviceSynchronize();
        CHECK_ERROR();

        // Allocate temporary storage
        sort_temp_storage = reinterpret_cast<void *>(gpuBufferManager->customCudaMalloc<uint8_t>(sort_temp_storage_bytes, 0, 0));

        // Run sorting operation
        hipcub::DeviceMergeSort::SortKeys(
            sort_temp_storage,
            sort_temp_storage_bytes,
            d_row_records,
            N,
            custom_less_comparator);

        hipDeviceSynchronize();
        CHECK_ERROR();
        auto sort_end_time = high_resolution_clock::now();
        auto sort_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(sort_end_time - sort_start_time).count();
        std::cout << "STRING GROUP BY V3: Sorting required " << sort_temp_storage_bytes << " bytes and took " << sort_time_ms << " ms" << std::endl;

        // Create a buffer of the aggregate values as well as an array of row ids
        uint64_t total_aggregation_bytes = 0;
        auto group_by_start_time = high_resolution_clock::now();
        uint64_t num_aggregate_values = N * (num_aggregates + 1);
        V* d_aggregate_buffer = gpuBufferManager->customCudaMalloc<V>(num_aggregate_values, 0, 0);
        uint64_t* d_aggregate_records = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
        total_aggregation_bytes += num_aggregate_values * sizeof(V) + N * sizeof(uint64_t);

        int *d_agg_mode = gpuBufferManager->customCudaMalloc<int>(num_aggregates, 0, 0);
        hipMemcpy(d_agg_mode, agg_mode, num_aggregates * sizeof(int), hipMemcpyHostToDevice);
        fill_aggregate_buffer<V><<<num_blocks, BLOCK_THREADS>>>(aggregate_keys, d_aggregate_buffer, d_aggregate_records, d_row_records,
                                                                d_agg_mode, N, num_aggregates);
        total_aggregation_bytes += num_aggregates * sizeof(int);

        // Create the additional fields we need to perform the group by
        sort_keys_type_string_v3* d_result_row_records = reinterpret_cast<sort_keys_type_string_v3 *>(gpuBufferManager->customCudaMalloc<pointer_and_two_values>(N, 0, 0));
        uint64_t* d_result_aggregate_records = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
        uint64_t* d_num_runs_out = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
        hipMemset(d_num_runs_out, 0, sizeof(uint64_t));
        CustomCombineOperatorV3<V> reduction_operation(d_aggregate_buffer, d_agg_mode, static_cast<int>(num_aggregates), N);

        // Now actually perform the group by
        void *d_group_by_temp_storage = nullptr;
        size_t group_by_temp_storage_bytes = 0;
        hipcub::DeviceReduce::ReduceByKey(
            d_group_by_temp_storage,
            group_by_temp_storage_bytes,
            d_row_records,
            d_result_row_records,
            d_aggregate_records,
            d_result_aggregate_records,
            d_num_runs_out,
            reduction_operation,
            N);

        // Allocate temporary storage
        d_group_by_temp_storage = gpuBufferManager->customCudaMalloc<uint8_t>(group_by_temp_storage_bytes, 0, 0);
        total_aggregation_bytes += group_by_temp_storage_bytes;

        hipcub::DeviceReduce::ReduceByKey(
            d_group_by_temp_storage,
            group_by_temp_storage_bytes,
            d_row_records,
            d_result_row_records,
            d_aggregate_records,
            d_result_aggregate_records,
            d_num_runs_out,
            reduction_operation,
            N);
        hipDeviceSynchronize();
        CHECK_ERROR();

        // Get the number of groups
        hipMemcpy(count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
        auto group_by_end_time = high_resolution_clock::now();
        auto group_by_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(group_by_end_time - group_by_start_time).count();
        std::cout << "STRING GROUP BY V3: Group By required " << total_aggregation_bytes << " bytes and took " << group_by_time_ms << " ms" << std::endl;

        auto post_processing_start_time = high_resolution_clock::now();
        
        // Create the vector of row_ids and update the aggregates for those rows to the aggregates for those groups
        uint64_t num_groups = count[0];
        uint64_t* d_group_row_ids = gpuBufferManager->customCudaMalloc<uint64_t>(num_groups, 0, 0);
        uint64_t num_group_blocks = (num_groups + BLOCK_THREADS - 1) / BLOCK_THREADS;
        
        perform_post_processing<<<num_group_blocks, BLOCK_THREADS>>>(d_result_aggregate_records, d_group_row_ids, aggregate_keys, 
            d_aggregate_buffer, d_agg_mode, num_groups, num_aggregates);
        hipDeviceSynchronize();
        CHECK_ERROR();

        // Materialize the string columns based on the row ids
        for(uint64_t i = 0; i < num_keys; i++) {
            // Get the original column
            uint8_t* group_key_chars = keys[i];
            uint64_t* group_key_offsets = offset[i];
            
            // Materailize the string column
            uint8_t* result; uint64_t* result_offset; uint64_t* new_num_bytes;
            materializeString(group_key_chars, group_key_offsets, result, result_offset, d_group_row_ids, new_num_bytes, num_groups);

            // Write back the result
            keys[i] = result;
            offset[i] = result_offset;
            num_bytes[i] = new_num_bytes[0];
        }
        
        auto post_processing_end_time = high_resolution_clock::now();
        auto post_processing_time_ms = std::chrono::duration_cast<duration<double, std::milli>>(post_processing_end_time - post_processing_start_time).count();
        std::cout << "STRING GROUP BY V3: Post Processing took " << post_processing_time_ms << " ms" << std::endl;
        
        std::cout << "STRING GROUP BY V3: Returning NDV of " << num_groups << std::endl;
    }

    template void groupedStringAggregateV3<double>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

    template void groupedStringAggregateV3<uint64_t>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t** offset, uint64_t* num_bytes, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

}