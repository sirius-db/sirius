#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#include "gpu_physical_grouped_aggregate.hpp"
#include "gpu_buffer_manager.hpp"

namespace duckdb {

struct sort_keys_type {
  uint64_t* keys;
  uint64_t num_key;

  __host__ __device__ sort_keys_type() {}
  __host__ __device__ sort_keys_type(uint64_t* keys, uint64_t num_key) : keys(keys), num_key(num_key) {}

  __host__ __device__ bool operator<(const sort_keys_type& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) {
            return keys[i] < other.keys[i];
        }
      }
      return true;
    }

    __host__ __device__ bool operator==(const sort_keys_type& other) const {
      for (uint64_t i = 0; i < num_key; i++) {
        if (keys[i] != other.keys[i]) return false;
      }
      return true;
    }
};

struct CustomLess
{
  __device__ bool operator()(const sort_keys_type &lhs, const sort_keys_type &rhs) {
      for (uint64_t i = 0; i < lhs.num_key; i++) {
            if (lhs.keys[i] != rhs.keys[i]) {
                return lhs.keys[i] < rhs.keys[i];
            }
      }
      return true;
  }
};

struct CustomSum
{
    template <typename T>
    __device__ T operator()(const T &a, const T &b) {
        return a + b;
    }
};

struct CustomMin
{
    template <typename T>
    __device__ T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};

struct CustomMax
{
    template <typename T>
    __device__ T operator()(const T &a, const T &b) const {
        return (b > a) ? b : a;
    }
};

template <typename T, int B, int I>
__global__ void columns_to_rows(T **a, T* result, sort_keys_type* temp, uint64_t N, uint64_t num_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            for (uint64_t i = 0; i < num_keys; i++) {
                result[offset * num_keys + i] = a[i][offset];
            }
            temp[offset] = sort_keys_type(&result[offset * num_keys], num_keys);
        }
    }
}

template <typename T, int B, int I>
__global__ void rows_to_columns(sort_keys_type *row_keys, T** col_keys, uint64_t N, uint64_t num_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            for (int i = 0; i < num_keys; i++) {
                // printf("Offset: %lu, Key[%d]: %lu\n", offset, i, row_keys[offset].keys[i]);
                col_keys[i][offset] = row_keys[offset].keys[i];
            }
        }
    }
}

template <typename T, int B, int I>
__global__ void gather_and_modify(const T *a, T* result, sort_keys_type *sort_keys, uint64_t N, uint64_t num_keys) {

    cudaAssert(num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            uint64_t items_ids = sort_keys[offset].keys[num_keys - 1];
            result[offset] = a[items_ids];
            sort_keys[offset] = sort_keys_type(sort_keys[offset].keys, num_keys - 1);
        }
    }
}

template <int B, int I>
__global__ void modify(sort_keys_type *sort_keys, uint64_t N, uint64_t num_keys) {

    cudaAssert(num_keys > 1);
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            uint64_t offset = tile_offset + threadIdx.x + ITEM * B;
            sort_keys[offset] = sort_keys_type(sort_keys[offset].keys, num_keys - 1);
        }
    }
}

template <int B, int I>
__global__ void sequence(uint64_t* result, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            result[tile_offset + threadIdx.x + ITEM * B] = tile_offset + threadIdx.x + ITEM * B;
        }
    }
}

template <typename T, int B, int I>
__global__ void divide(T* a, uint64_t* b, T* result, uint64_t N) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            int offset = tile_offset + threadIdx.x + ITEM * B;
            result[offset] = a[offset] / b[offset];
        }
    }
}

template <typename T, int B, int I>
__global__ void fill_n(T* a, T b, uint64_t N) {
    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            a[tile_offset + threadIdx.x + ITEM * B] = b;
        }
    }
}

template <typename T>
__global__ void testprint(T* a, uint64_t N) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (uint64_t i = 0; i < 100; i++) {
            printf("%.2f ", a[i]);
        }
        printf("\n");
    }
}

template
__global__ void gather_and_modify<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(const uint64_t *a, uint64_t* result, sort_keys_type* sort_keys, uint64_t N, uint64_t num_keys);
template
__global__ void gather_and_modify<double, BLOCK_THREADS, ITEMS_PER_THREAD>(const double *a, double* result, sort_keys_type* sort_keys, uint64_t N, uint64_t num_keys);

template
__global__ void columns_to_rows<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t **a, uint64_t* result, sort_keys_type* temp, uint64_t N, uint64_t num_keys);

template
__global__ void rows_to_columns<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD>(sort_keys_type *row_keys, uint64_t** col_keys, uint64_t N, uint64_t num_keys);

template <typename T, typename V>
void groupedAggregate(uint8_t **keys, uint8_t **aggregate_keys, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode) {
    CHECK_ERROR();
    if (N == 0) {
        count[0] = 0;
        printf("N is 0\n");
        return;
    }

    printf("Launching Grouped Aggregate Kernel\n");
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    //allocate temp memory and copying keys
    T* row_keys = gpuBufferManager->customCudaMalloc<T>((num_keys + 1) * N, 0, 0).data_;
    sort_keys_type* materialized_temp = reinterpret_cast<sort_keys_type*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0).data_);
    // T* keys_row_id[num_keys + 1];
    T** keys_row_id = new T*[num_keys + 1];
    for (uint64_t i = 0; i < num_keys; i++) {
        keys_row_id[i] = reinterpret_cast<T*> (keys[i]);
    }

    //generate sequence
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    uint64_t* row_sequence = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
    sequence<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(row_sequence, N);
    keys_row_id[num_keys] = row_sequence;

    T** keys_dev;
    hipMalloc((void**) &keys_dev, (num_keys + 1) * sizeof(T*));
    hipMemcpy(keys_dev, keys_row_id, (num_keys + 1) * sizeof(T*), hipMemcpyHostToDevice);

    columns_to_rows<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, row_keys, materialized_temp, N, num_keys + 1);
    CHECK_ERROR();
    hipDeviceSynchronize();

    //perform sort-based groupby
    // Determine temporary device storage requirements
    CustomLess custom_less;
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    // Allocate temporary storage
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

    // Run sorting operation
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    //gather the aggregates based on the row_sequence
    printf("Gathering Aggregates\n");
    V** aggregate_keys_temp = new V*[num_aggregates];
    uint64_t** aggregate_star_temp = new uint64_t*[num_aggregates];
    sort_keys_type* group_by_rows = reinterpret_cast<sort_keys_type*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0).data_);
    uint64_t* d_num_runs_out = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0).data_;
    hipMemset(d_num_runs_out, 0, sizeof(uint64_t));
    uint64_t* h_count = new uint64_t[1];

    for (int agg = 0; agg < num_aggregates; agg++) {
        // printf("Aggregating %d\n", agg);
        if (agg_mode[agg] == 4 || agg_mode[agg] == 5) { //count_star or count(null) or sum(null)
            aggregate_star_temp[agg] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
            if (agg_mode[agg] == 4) 
                fill_n<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(aggregate_star_temp[agg], 1, N);
            else if (agg_mode[agg] == 5)
                hipMemset(aggregate_star_temp[agg], 0, N * sizeof(uint64_t));

            modify<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(materialized_temp, N, num_keys + 1);

            //perform reduce_by_key
            uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
            hipMemset(agg_star_out, 0, N * sizeof(uint64_t));

            printf("Reduce by key count_star\n");
            // Determine temporary device storage requirements
            d_temp_storage = nullptr;
            temp_storage_bytes = 0;
            CustomSum custom_sum;
            hipcub::DeviceReduce::ReduceByKey(
                d_temp_storage, temp_storage_bytes,
                materialized_temp, group_by_rows, aggregate_star_temp[agg],
                agg_star_out, d_num_runs_out, custom_sum, N);

            CHECK_ERROR();

            // Allocate temporary storage
            d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

            // Run reduce-by-key
            hipcub::DeviceReduce::ReduceByKey(
                d_temp_storage, temp_storage_bytes,
                materialized_temp, group_by_rows, aggregate_star_temp[agg],
                agg_star_out, d_num_runs_out, custom_sum, N);

            CHECK_ERROR();

            hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
            count[0] = h_count[0];

            printf("Count: %lu\n", count[0]);

            CHECK_ERROR();
            aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_star_out);
        } else {
            aggregate_keys_temp[agg] = gpuBufferManager->customCudaMalloc<V>(N, 0, 0).data_;
            V* temp = reinterpret_cast<V*> (aggregate_keys[agg]);
            gather_and_modify<V, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(temp, aggregate_keys_temp[agg], materialized_temp, N, num_keys + 1);

            V* agg_out = gpuBufferManager->customCudaMalloc<V>(N, 0, 0).data_;
            hipMemset(agg_out, 0, N * sizeof(V));

            CHECK_ERROR();
            if (agg_mode[agg] == 0) {
                printf("Reduce by key sum\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomSum custom_sum;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_out);
            } else if (agg_mode[agg] == 1) {
                //Currently typename V has to be a double
                printf("Reduce by key avg\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomSum custom_sum;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                aggregate_star_temp[agg] = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
                fill_n<uint64_t, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(aggregate_star_temp[agg], 1, N);

                uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
                hipMemset(agg_star_out, 0, N * sizeof(uint64_t));
                hipMemset(d_num_runs_out, 0, sizeof(uint64_t));

                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_star_temp[agg],
                    agg_star_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_star_temp[agg],
                    agg_star_out, d_num_runs_out, custom_sum, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                V* output = gpuBufferManager->customCudaMalloc<V>(count[0], 0, 0).data_;
                divide<V, BLOCK_THREADS, ITEMS_PER_THREAD><<<(count[0] + tile_items - 1)/tile_items, BLOCK_THREADS>>>(agg_out, agg_star_out, output, count[0]);

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (output);
            } else if (agg_mode[agg] == 2) {
                printf("Reduce by key max\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomMax custom_max;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_max, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_max, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_out);
            } else if (agg_mode[agg] == 3) {
                printf("Reduce by key min\n");
                // Determine temporary device storage requirements
                d_temp_storage = nullptr;
                temp_storage_bytes = 0;
                CustomMin custom_min;
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_min, N);

                CHECK_ERROR();

                // Allocate temporary storage
                d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

                // Run reduce-by-key
                hipcub::DeviceReduce::ReduceByKey(
                    d_temp_storage, temp_storage_bytes,
                    materialized_temp, group_by_rows, aggregate_keys_temp[agg],
                    agg_out, d_num_runs_out, custom_min, N);

                CHECK_ERROR();

                hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
                count[0] = h_count[0];

                CHECK_ERROR();
                aggregate_keys[agg] = reinterpret_cast<uint8_t*> (agg_out);
            }
        }
    }

    T** keys_dev_result;
    T** keys_result = new T*[num_keys];
    hipMalloc((void**) &keys_dev_result, num_keys * sizeof(T*));
    for (uint64_t i = 0; i < num_keys; i++) {
        keys_result[i] = gpuBufferManager->customCudaMalloc<T>(count[0], 0, 0).data_;
    }
    hipMemcpy(keys_dev_result, keys_result, num_keys * sizeof(T*), hipMemcpyHostToDevice);

    rows_to_columns<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(count[0] + tile_items - 1)/tile_items, BLOCK_THREADS>>>(group_by_rows, keys_dev_result, count[0], num_keys);

    CHECK_ERROR();
    hipDeviceSynchronize();
    printf("Count: %lu\n", count[0]);

    for (uint64_t i = 0; i < num_keys; i++) {
        keys[i] = reinterpret_cast<uint8_t*> (keys_result[i]);
    }
}


template <typename T>
void groupedWithoutAggregate(uint8_t **keys, uint64_t* count, uint64_t N, uint64_t num_keys) {
    CHECK_ERROR();
    if (N == 0) {
        count[0] = 0;
        printf("N is 0\n");
        return;
    }
    printf("Launching Grouped Without Aggregate Kernel\n");
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    //allocate temp memory and copying keys
    T* row_keys = gpuBufferManager->customCudaMalloc<T>(num_keys * N, 0, 0).data_;
    sort_keys_type* materialized_temp = reinterpret_cast<sort_keys_type*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0).data_);
    T** keys_row_id = new T*[num_keys];
    for (uint64_t i = 0; i < num_keys; i++) {
        keys_row_id[i] = reinterpret_cast<T*> (keys[i]);
    }

    //generate sequence
    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    T** keys_dev;
    hipMalloc((void**) &keys_dev, num_keys * sizeof(T*));
    hipMemcpy(keys_dev, keys_row_id, num_keys * sizeof(T*), hipMemcpyHostToDevice);

    columns_to_rows<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, row_keys, materialized_temp, N, num_keys);
    CHECK_ERROR();

    //perform sort-based groupby
    // Determine temporary device storage requirements
    CustomLess custom_less;
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    // Allocate temporary storage
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

    // Run sorting operation
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        materialized_temp,
        N,
        custom_less);

    CHECK_ERROR();

    //gather the aggregates based on the row_sequence
    // printf("Gathering Aggregates\n");
    sort_keys_type* group_by_rows = reinterpret_cast<sort_keys_type*> (gpuBufferManager->customCudaMalloc<pointer_and_key>(N, 0, 0).data_);
    uint64_t* d_num_runs_out = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0).data_;
    hipMemset(d_num_runs_out, 0, sizeof(uint64_t));
    uint64_t* h_count = new uint64_t[1];

    uint64_t* aggregate_star_temp = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
    hipMemset(aggregate_star_temp, 0, N * sizeof(uint64_t));

    //perform reduce_by_key
    uint64_t* agg_star_out = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0).data_;
    hipMemset(agg_star_out, 0, N * sizeof(uint64_t));

    // printf("Reduce by key count_star\n");
    // Determine temporary device storage requirements
    d_temp_storage = nullptr;
    temp_storage_bytes = 0;
    CustomSum custom_sum;
    hipcub::DeviceReduce::ReduceByKey(
        d_temp_storage, temp_storage_bytes,
        materialized_temp, group_by_rows, aggregate_star_temp,
        agg_star_out, d_num_runs_out, custom_sum, N);

    CHECK_ERROR();

    // Allocate temporary storage
    d_temp_storage = reinterpret_cast<void*> (gpuBufferManager->customCudaMalloc<uint8_t>(temp_storage_bytes, 0, 0).data_);

    // Run reduce-by-key
    hipcub::DeviceReduce::ReduceByKey(
        d_temp_storage, temp_storage_bytes,
        materialized_temp, group_by_rows, aggregate_star_temp,
        agg_star_out, d_num_runs_out, custom_sum, N);

    CHECK_ERROR();

    hipMemcpy(h_count, d_num_runs_out, sizeof(uint64_t), hipMemcpyDeviceToHost);
    count[0] = h_count[0];

    T** keys_dev_result;
    T** keys_result = new T*[num_keys];
    hipMalloc((void**) &keys_dev_result, num_keys * sizeof(T*));
    for (uint64_t i = 0; i < num_keys; i++) {
        keys_result[i] = gpuBufferManager->customCudaMalloc<T>(count[0], 0, 0).data_;
    }
    hipMemcpy(keys_dev_result, keys_result, num_keys * sizeof(T*), hipMemcpyHostToDevice);

    rows_to_columns<T, BLOCK_THREADS, ITEMS_PER_THREAD><<<(count[0] + tile_items - 1)/tile_items, BLOCK_THREADS>>>(group_by_rows, keys_dev_result, count[0], num_keys);

    CHECK_ERROR();
    hipDeviceSynchronize();
    printf("Count: %lu\n", count[0]);

    for (uint64_t i = 0; i < num_keys; i++) {
        keys[i] = reinterpret_cast<uint8_t*> (keys_result[i]);
    }
}

template<typename T>
void combineColumns(T* a, T* b, T* c, uint64_t N_a, uint64_t N_b) {
    CHECK_ERROR();
    if (N_a == 0 || N_b == 0) {
        printf("N is 0\n");
        return;
    }
    hipMemcpy(c, a, N_a * sizeof(T), hipMemcpyDeviceToDevice);
    hipMemcpy(c + N_a, b, N_b * sizeof(T), hipMemcpyDeviceToDevice);
    CHECK_ERROR();
    hipDeviceSynchronize();
}

template
void groupedAggregate<uint64_t, uint64_t>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

template
void groupedAggregate<uint64_t, double>(uint8_t **keys, uint8_t **aggregate_keys, uint64_t* count, uint64_t N, uint64_t num_keys, uint64_t num_aggregates, int* agg_mode);

template
void groupedWithoutAggregate<uint64_t>(uint8_t **keys, uint64_t* count, uint64_t N, uint64_t num_keys);

template
void combineColumns<uint64_t>(uint64_t* a, uint64_t* b, uint64_t* c, uint64_t N_a, uint64_t N_b);

template
void combineColumns<double>(double* a, double* b, double* c, uint64_t N_a, uint64_t N_b);

}