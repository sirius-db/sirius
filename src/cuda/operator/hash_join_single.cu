#include "hip/hip_runtime.h"
/*
 * Copyright 2025, Sirius Contributors.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_helper.cuh"
#include "gpu_physical_hash_join.hpp"
#include "gpu_buffer_manager.hpp"
#include "log/logging.hpp"

namespace duckdb {

__device__ uint64_t hash64(uint64_t key1, uint64_t key2) {
    uint64_t h = key1 * 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    h ^= key2 * 0xc6a4a7935bd1e995ull;
    h *= 0xc6a4a7935bd1e995ull;
    h ^= (h >> 33);
    return h;
}

template <int B, int I, typename T>
__global__ void probe_right_semi_anti_single(T **keys, unsigned long long* ht, uint64_t ht_len,
            uint64_t N, int* condition_mode, int num_keys, int equal_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            
            uint64_t slot;
            if (equal_keys == 1) slot = keys[0][tile_offset + threadIdx.x + ITEM * B] % ht_len;
            else if (equal_keys == 2) slot = hash64(keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B]) % ht_len;
            else cudaAssert(0);
            
            while (ht[slot * (num_keys + 2)] != 0xFFFFFFFFFFFFFFFF) {
                bool local_found = 1;
                for (int n = 0; n < num_keys; n++) {
                    uint64_t item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0 && ht[slot * (num_keys + 2) + n] != item) local_found = 0;
                    else if (condition_mode[n] == 1 && ht[slot * (num_keys + 2) + n] == item) local_found = 0;
                }
                if (local_found) {
                    ht[slot * (num_keys + 2) + num_keys + 1] = tile_offset + threadIdx.x + ITEM * B;
                    break;
                }
                slot = (slot + 100007) % ht_len;
            }
        }
    }
}

template <int B, int I, typename T>
__global__ void probe_single_match(T **keys, unsigned long long* ht, uint64_t ht_len, uint64_t *row_ids_left, uint64_t *row_ids_right, unsigned long long* count, 
            uint64_t N, int* condition_mode, int num_keys, int equal_keys, int join_mode, bool is_count) {

    typedef hipcub::BlockScan<int, B> BlockScanInt;

    __shared__ union TempStorage
    {
        typename BlockScanInt::TempStorage scan;
    } temp_storage;

    int items_off[I];
    int selection_flags[I];

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    int t_count = 0; // Number of items selected per thread
    int c_t_count = 0; //Prefix sum of t_count
    __shared__ uint64_t block_off;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        selection_flags[ITEM] = 0;
    }

    // int n_ht_column = num_keys + 1;
    int n_ht_column;
    if (join_mode == 3) n_ht_column = num_keys + 2;
    else n_ht_column = num_keys + 1;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            
            uint64_t slot;
            if (equal_keys == 1) slot = keys[0][tile_offset + threadIdx.x + ITEM * B] % ht_len;
            else if (equal_keys == 2) slot = hash64(keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B]) % ht_len;
            else cudaAssert(0);
            
            bool found = 0;
            while (ht[slot * n_ht_column] != 0xFFFFFFFFFFFFFFFF) {
                bool local_found = 1;
                for (int n = 0; n < num_keys; n++) {
                    uint64_t item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0 && ht[slot * n_ht_column + n] != item) local_found = 0;
                    else if (condition_mode[n] == 1 && ht[slot * n_ht_column + n] == item) local_found = 0;
                }
                if (local_found) {
                    items_off[ITEM] = ht[slot * n_ht_column + num_keys];
                    found = 1;
                    break;
                }
                slot = (slot + 100007) % ht_len;
            }

            if (join_mode == 2) { // anti join
                if (!found) {
                    t_count++;
                    selection_flags[ITEM] = 1;
                }
            } else {
                if (found) {
                    if (join_mode == 3) ht[slot * (num_keys + 2) + num_keys + 1] = tile_offset + threadIdx.x + ITEM * B;
                    t_count++;
                    selection_flags[ITEM] = 1;
                }
            }
        }
    }

    //Barrier
    __syncthreads();

    BlockScanInt(temp_storage.scan).ExclusiveSum(t_count, c_t_count); //doing a prefix sum of all the previous threads in the block and store it to c_t_count
    if(threadIdx.x == blockDim.x - 1) { //if the last thread in the block, add the prefix sum of all the prev threads + sum of my threads to global variable total
        block_off = atomicAdd(count, (unsigned long long) t_count+c_t_count); //the previous value of total is gonna be assigned to block_off
    } //block_off does not need to be global (it's just need to be shared), because it will get the previous value from total which is global

    __syncthreads();

    if (is_count) return;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ++ITEM) {
        if (threadIdx.x + ITEM * B < num_tile_items) {
            if(selection_flags[ITEM]) {
                uint64_t offset = block_off + c_t_count++;
                if (join_mode == 0 || join_mode == 3) { // inner join and right join
                    row_ids_right[offset] = items_off[ITEM];
                    row_ids_left[offset] = tile_offset + threadIdx.x + ITEM * B;
                } else if (join_mode == 1 || join_mode == 2) { // semi join and anti join
                    row_ids_left[offset] = tile_offset + threadIdx.x + ITEM * B;
                } else {
                    cudaAssert(0);
                }
            }
        }
    }

}

template <int B, int I>
__global__ void probe_mark(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint8_t* output,
            uint64_t N, int* condition_mode, int num_keys, int equal_keys) {

    uint64_t tile_size = B * I;
    uint64_t tile_offset = blockIdx.x * tile_size;

    uint64_t num_tiles = (N + tile_size - 1) / tile_size;
    uint64_t num_tile_items = tile_size;

    if (blockIdx.x == num_tiles - 1) {
        num_tile_items = N - tile_offset;
    }

    int n_ht_column = num_keys + 1;

    #pragma unroll
    for (int ITEM = 0; ITEM < I; ITEM++) {
        if (threadIdx.x + (ITEM * B) < num_tile_items) {
            
            uint64_t slot;
            if (equal_keys == 1) slot = keys[0][tile_offset + threadIdx.x + ITEM * B] % ht_len;
            else if (equal_keys == 2) slot = hash64(keys[0][tile_offset + threadIdx.x + ITEM * B], keys[1][tile_offset + threadIdx.x + ITEM * B]) % ht_len;
            else cudaAssert(0);
            
            bool found = 0;
            while (ht[slot * n_ht_column] != 0xFFFFFFFFFFFFFFFF) {
                bool local_found = 1;
                for (int n = 0; n < num_keys; n++) {
                    uint64_t item = keys[n][tile_offset + threadIdx.x + ITEM * B];
                    if (condition_mode[n] == 0 && ht[slot * n_ht_column + n] != item) local_found = 0;
                    else if (condition_mode[n] == 1 && ht[slot * n_ht_column + n] == item) local_found = 0;
                }
                if (local_found) {
                    found = 1;
                    break;
                }
                slot = (slot + 100007) % ht_len;
            }

            output[tile_offset + threadIdx.x + ITEM * B] = found;
        }
    }
}

template
__global__ void probe_mark<BLOCK_THREADS, ITEMS_PER_THREAD>(uint64_t **keys, unsigned long long* ht, uint64_t ht_len, uint8_t* output,
            uint64_t N, int* condition_mode, int num_keys, int equal_keys);

template <typename T>
void probeHashTableSingleMatch(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t* &row_ids_left, uint64_t* &row_ids_right, 
            uint64_t* &count, uint64_t N, int* condition_mode, int num_keys, int join_mode) {
    CHECK_ERROR();
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    if (N == 0 || ht_len == 0) {
        uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
        h_count[0] = 0;
        count = h_count;
        SIRIUS_LOG_DEBUG("Input size is 0 or hash table is empty");
        return;
    }
    SIRIUS_LOG_DEBUG("Launching Probe Kernel Unique Join");
    SETUP_TIMING();
    START_TIMER();
    count = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
    hipMemset(count, 0, sizeof(uint64_t));

    //reinterpret cast the keys to type T
    T** keys_data = gpuBufferManager->customCudaHostAlloc<T*>(num_keys);
    for (int idx = 0; idx < num_keys; idx++) {
        keys_data[idx] = reinterpret_cast<T*>(keys[idx]);
    }

    T** keys_dev = gpuBufferManager->customCudaMalloc<T*>(num_keys, 0, 0);
    hipMemcpy(keys_dev, keys_data, num_keys * sizeof(T*), hipMemcpyHostToDevice);

    int equal_keys = 0;
    for (int idx = 0; idx < num_keys; idx++) {
        if (condition_mode[idx] == 0) equal_keys++;
    }

    int* condition_mode_dev = gpuBufferManager->customCudaMalloc<int>(num_keys, 0, 0);
    hipMemcpy(condition_mode_dev, condition_mode, num_keys * sizeof(int), hipMemcpyHostToDevice);

    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;

    // size_t openmalloc_full = (gpuBufferManager->processing_size_per_gpu - gpuBufferManager->gpuProcessingPointer[0] - 1024) / sizeof(uint64_t);
    // size_t openmalloc_half = openmalloc_full / 2;
    // row_ids_left = gpuBufferManager->customCudaMalloc<uint64_t>(openmalloc_half, 0, 0);
    row_ids_left = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
    // if (join_mode == 0 || join_mode == 3) row_ids_right = gpuBufferManager->customCudaMalloc<uint64_t>(openmalloc_half, 0, 0);
    if (join_mode == 0 || join_mode == 3) row_ids_right = gpuBufferManager->customCudaMalloc<uint64_t>(N, 0, 0);
    hipMemset(count, 0, sizeof(uint64_t));
    probe_single_match<BLOCK_THREADS, ITEMS_PER_THREAD, T><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, ht, ht_len, row_ids_left, row_ids_right, (unsigned long long*) count, 
            N, condition_mode_dev, num_keys, equal_keys, join_mode, 0);
    CHECK_ERROR();
    hipDeviceSynchronize();

    uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
    hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
    assert(h_count[0] > 0);
    SIRIUS_LOG_DEBUG("Probe Hash Table Single Match Result Count: {}", h_count[0]);
    // if (join_mode == 0 || join_mode == 3) {
    //     gpuBufferManager->gpuProcessingPointer[0] = (reinterpret_cast<uint8_t*>(row_ids_left + h_count[0]) - gpuBufferManager->gpuProcessing[0]);
    //     cudaMemmove(reinterpret_cast<uint8_t*>(row_ids_left + h_count[0]), reinterpret_cast<uint8_t*>(row_ids_right), h_count[0] * sizeof(uint64_t));
    //     CHECK_ERROR();
    //     row_ids_right = row_ids_left + h_count[0];
    //     gpuBufferManager->gpuProcessingPointer[0] = (reinterpret_cast<uint8_t*>(row_ids_right + h_count[0]) - gpuBufferManager->gpuProcessing[0]);
    // } else {
    //     gpuBufferManager->gpuProcessingPointer[0] = (reinterpret_cast<uint8_t*>(row_ids_left + h_count[0]) - gpuBufferManager->gpuProcessing[0]);
    // }

    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(keys_dev), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(count), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(condition_mode_dev), 0);

    count = h_count;
    STOP_TIMER();
}

template <typename T>
void probeHashTableRightSemiAntiSingleMatch(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N, int* condition_mode, int num_keys) {
    CHECK_ERROR();
    if (N == 0 || ht_len == 0) {
        SIRIUS_LOG_DEBUG("Input size is 0 or hash table is empty");
        return;
    }
    SIRIUS_LOG_DEBUG("Launching Probe Kernel Unique Join");
    SETUP_TIMING();
    START_TIMER();
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());

    //reinterpret cast the keys to type T
    T** keys_data = gpuBufferManager->customCudaHostAlloc<T*>(num_keys);
    for (int idx = 0; idx < num_keys; idx++) {
        keys_data[idx] = reinterpret_cast<T*>(keys[idx]);
    }

    T** keys_dev = gpuBufferManager->customCudaMalloc<T*>(num_keys, 0, 0);
    hipMemcpy(keys_dev, keys_data, num_keys * sizeof(T*), hipMemcpyHostToDevice);

    int equal_keys = 0;
    for (int idx = 0; idx < num_keys; idx++) {
        if (condition_mode[idx] == 0) equal_keys++;
    }

    int* condition_mode_dev = gpuBufferManager->customCudaMalloc<int>(num_keys, 0, 0);
    hipMemcpy(condition_mode_dev, condition_mode, num_keys * sizeof(int), hipMemcpyHostToDevice);

    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    probe_right_semi_anti_single<BLOCK_THREADS, ITEMS_PER_THREAD, T><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, ht, ht_len, N, condition_mode_dev, num_keys, equal_keys);
    CHECK_ERROR();
    hipDeviceSynchronize();

    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(keys_dev), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(condition_mode_dev), 0);

    SIRIUS_LOG_DEBUG("Finished probe right");
    STOP_TIMER();
}

void probeHashTableMark(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint8_t* &output, uint64_t N, int* condition_mode, int num_keys) {
    CHECK_ERROR();
    GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
    if (N == 0 || ht_len == 0) {
        output = gpuBufferManager->customCudaMalloc<uint8_t>(N, 0, 0);
        hipMemset(output, 0, N * sizeof(uint8_t));
        SIRIUS_LOG_DEBUG("Input size is 0 or hash table is empty");
        return;
    }
    SIRIUS_LOG_DEBUG("Launching Probe Kernel Mark");
    SETUP_TIMING();
    START_TIMER();

    //reinterpret cast the keys to uint64_t
    uint64_t** keys_data = gpuBufferManager->customCudaHostAlloc<uint64_t*>(num_keys);
    for (int idx = 0; idx < num_keys; idx++) {
        keys_data[idx] = reinterpret_cast<uint64_t*>(keys[idx]);
    }

    uint64_t** keys_dev = gpuBufferManager->customCudaMalloc<uint64_t*>(num_keys, 0, 0);
    hipMemcpy(keys_dev, keys_data, num_keys * sizeof(uint64_t*), hipMemcpyHostToDevice);

    CHECK_ERROR();

    int equal_keys = 0;
    for (int idx = 0; idx < num_keys; idx++) {
        if (condition_mode[idx] == 0) equal_keys++;
    }

    int* condition_mode_dev = gpuBufferManager->customCudaMalloc<int>(num_keys, 0, 0);
    hipMemcpy(condition_mode_dev, condition_mode, num_keys * sizeof(int), hipMemcpyHostToDevice);
    output = gpuBufferManager->customCudaMalloc<uint8_t>(N, 0, 0);

    int tile_items = BLOCK_THREADS * ITEMS_PER_THREAD;
    CHECK_ERROR();
    probe_mark<BLOCK_THREADS, ITEMS_PER_THREAD><<<(N + tile_items - 1)/tile_items, BLOCK_THREADS>>>(keys_dev, ht, ht_len, output, 
            N, condition_mode_dev, num_keys, equal_keys);
    CHECK_ERROR();
    hipDeviceSynchronize();
    STOP_TIMER();

    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(keys_dev), 0);
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(condition_mode_dev), 0);
}

template
void probeHashTableSingleMatch<int32_t>(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t* &row_ids_left, uint64_t* &row_ids_right, 
            uint64_t* &count, uint64_t N, int* condition_mode, int num_keys, int join_mode);

template
void probeHashTableSingleMatch<int64_t>(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t* &row_ids_left, uint64_t* &row_ids_right, 
            uint64_t* &count, uint64_t N, int* condition_mode, int num_keys, int join_mode);

template
void probeHashTableRightSemiAntiSingleMatch<int32_t>(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N, int* condition_mode, int num_keys);

template
void probeHashTableRightSemiAntiSingleMatch<int64_t>(uint8_t **keys, unsigned long long* ht, uint64_t ht_len, uint64_t N, int* condition_mode, int num_keys);

}