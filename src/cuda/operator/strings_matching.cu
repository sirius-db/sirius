#include "hip/hip_runtime.h"
/*
 * Copyright 2025, Sirius Contributors.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cuda_helper.cuh"
#include "cuda_stream_view.hpp"
#include "gpu_physical_strings_matching.hpp"
#include "gpu_buffer_manager.hpp"
#include "log/logging.hpp"

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

#define THREADS_PER_BLOCK_STRINGS 512
#define WARP_SIZE 32
#define CHARS_IN_BYTE 256
#define CHAR_INCREMENT 128
#define INITIAL_MEMORY_FACTOR 2.0
#define CHUNK_SIZE 8192
#define TILE_ITEMS_PER_TILE 10

namespace duckdb {

//--------------------------------------------------//
// String Matching
//--------------------------------------------------//
template<typename IdxT>
__global__ void determine_start_kernel(const IdxT* indices, IdxT num_strings, IdxT* worker_start_term, IdxT num_workers, IdxT chunk_size, IdxT last_char) {  
  IdxT tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= num_workers) return; 

  IdxT curr_chunk_start = min(tid * chunk_size, last_char);
  IdxT search_start_term = 0; IdxT search_end_term = num_strings;
  IdxT curr_worker_start = 0; // TODO: CHECK IT'S OKAY TO INITIALIZE IT TO 0
  // int curr_worker_start = -1;
  IdxT curr_search_term;
  while(search_start_term <= search_end_term) {
    curr_search_term = (search_start_term + search_end_term)/2;

    // Determine if this workers chunk is in this terms range
    if(curr_chunk_start >= indices[curr_search_term] && curr_chunk_start < indices[curr_search_term + 1]) {
        curr_worker_start = curr_search_term;
        break;
    } else if(curr_chunk_start < indices[curr_search_term]) {
        // The chunk starts before this term so search lower range
        search_end_term = curr_search_term - 1;
    } else {
        // The chunk starts after this term so search an upper range
        search_start_term = curr_search_term + 1;
    }
  }

  worker_start_term[tid] = curr_worker_start;
}

// Instantiations
template __global__ void determine_start_kernel<uint64_t>(const uint64_t* indices,
                                                          uint64_t num_strings,
                                                          uint64_t* worker_start_term,
                                                          uint64_t num_workers,
                                                          uint64_t chunk_size,
                                                          uint64_t last_char);
template __global__ void determine_start_kernel<int64_t>(const int64_t* indices,
                                                         int64_t num_strings,
                                                         int64_t* worker_start_term,
                                                         int64_t num_workers,
                                                         int64_t chunk_size,
                                                         int64_t last_char);

template<typename IdxT>
__global__ void single_term_kmp_kernel(const char* char_data, const IdxT* indices, const int* kmp_automato, const IdxT* worker_start_term, bool* results, 
IdxT pattern_size, IdxT num_workers, IdxT chunk_size, IdxT sub_chunk_size, IdxT last_char, IdxT num_strings) {
    
    // See if have any work to do
    auto chunk_id = blockIdx.x;
    if (chunk_id >= num_workers) return;

    const auto curr_chunk_start = min(chunk_id * chunk_size, last_char);
    const auto curr_chunk_end = min(curr_chunk_start + chunk_size + pattern_size, last_char);
    const auto curr_sub_chunk_start = min(curr_chunk_start + threadIdx.x * sub_chunk_size, curr_chunk_end);
    const auto curr_sub_chunk_end = min(curr_sub_chunk_start + sub_chunk_size + pattern_size, curr_chunk_end);

    // Determine the subchunk that the current string is going to be working on
    auto curr_term = worker_start_term[chunk_id];
    while (curr_term < num_strings && (curr_sub_chunk_start < indices[curr_term] || curr_sub_chunk_start >= indices[curr_term + 1])) {
      curr_term++;
    }
    auto curr_term_end = indices[curr_term + 1];

    // Perform the actual string matching
    int j = 0; int curr_idx = 0; 
    #pragma unroll
    for(int i = curr_sub_chunk_start; i <= curr_sub_chunk_end; i++) {
        // See if we need to switch to a new term
        if(i >= curr_term_end) {
          curr_term = curr_term + 1;
          curr_term_end = indices[curr_term + 1];
          j = 0; // Reset because we are at the start of the string
        }

        curr_idx = (int) char_data[i] + CHAR_INCREMENT;
        j = kmp_automato[j * CHARS_IN_BYTE + curr_idx];

        // Record that we have a hit
        if(j >= pattern_size) {
          results[curr_term] = true;
          j = 0;
        }
    }
}

// Instantiations
template __global__ void single_term_kmp_kernel<uint64_t>(const char* char_data,
                                                          const uint64_t* indices,
                                                          const int* kmp_automato,
                                                          const uint64_t* worker_start_term,
                                                          bool* results,
                                                          uint64_t pattern_size,
                                                          uint64_t num_workers,
                                                          uint64_t chunk_size,
                                                          uint64_t sub_chunk_size,
                                                          uint64_t last_char,
                                                          uint64_t num_strings);
template __global__ void single_term_kmp_kernel<int64_t>(const char* char_data,
                                                         const int64_t* indices,
                                                         const int* kmp_automato,
                                                         const int64_t* worker_start_term,
                                                         bool* results,
                                                         int64_t pattern_size,
                                                         int64_t num_workers,
                                                         int64_t chunk_size,
                                                         int64_t sub_chunk_size,
                                                         int64_t last_char,
                                                         int64_t num_strings);

__global__ void write_matching_rows(bool* results, uint64_t num_strings, uint64_t* matching_rows, uint64_t* count) {
  uint64_t tile_size = gridDim.x * blockDim.x;
  uint64_t start_idx = threadIdx.x + blockIdx.x * blockDim.x;
  for(uint64_t i = start_idx; i < num_strings; i += tile_size) {
    if(results[i]) {
      uint64_t write_offset = atomicAdd(reinterpret_cast<unsigned long long int*>(count), 1);
      matching_rows[write_offset] = i;
    }
  }
}

void StringMatching(char* char_data, uint64_t* str_indices, std::string match_string, uint64_t* &row_id, uint64_t* &count, uint64_t num_chars, uint64_t num_strings, int not_equal) {
  CHECK_ERROR();
  GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
  if (num_strings == 0) {
    SIRIUS_LOG_DEBUG("Input size is 0");
    uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
    h_count[0] = 0;
    count = h_count;
    return;
  }

    SETUP_TIMING();
    START_TIMER();
  SIRIUS_LOG_DEBUG("Launching single term string matching kernel");
  // Get the data from the metadata
  uint64_t workers_needed = (num_chars + CHUNK_SIZE - 1)/CHUNK_SIZE;

  // Compute the automato for this string
  const int match_length = match_string.size();
  const char* match_char = match_string.c_str();
  int kmp_automato_size = match_length * CHARS_IN_BYTE;
  int* kmp_automato = gpuBufferManager->customCudaHostAlloc<int>(kmp_automato_size);
  std::memset(kmp_automato, 0, kmp_automato_size * sizeof(int));
  int first_idx = (int) match_char[0] + CHAR_INCREMENT;
  kmp_automato[first_idx] = 1;
  for(int X = 0, j = 1; j < match_length; j++) {
    int curr_idx = (int) match_char[j] + CHAR_INCREMENT;

    // Copy over the chars from the previous automato
    for(int c = 0; c < CHARS_IN_BYTE; c++) {
        kmp_automato[j * CHARS_IN_BYTE + c] = kmp_automato[X * CHARS_IN_BYTE + c];
    }
    kmp_automato[j * CHARS_IN_BYTE + curr_idx] = j + 1;
    X = kmp_automato[X * CHARS_IN_BYTE + curr_idx];
  }

  // Allocate the buffers we need
  count = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
  char* d_match_str = gpuBufferManager->customCudaMalloc<char>(match_string.length(), 0, 0);
  int* d_kmp_automato = gpuBufferManager->customCudaMalloc<int>(kmp_automato_size, 0, 0);
  uint64_t* d_worker_start_term = gpuBufferManager->customCudaMalloc<uint64_t>(workers_needed, 0, 0);
  bool* d_answers = reinterpret_cast<bool*> (gpuBufferManager->customCudaMalloc<uint8_t>(num_strings, 0, 0));
  hipMemset(d_answers, 0, num_strings * sizeof(bool));
  // TODO: Do it twice for more accurate allocation

  // Copy over the data to the buffers
  hipMemcpy(d_kmp_automato, kmp_automato, kmp_automato_size * sizeof(int), hipMemcpyHostToDevice);

  // Also set the initial values
  // hipMemset(d_matching_rows, 0, num_strings * sizeof(uint64_t));
  CHECK_ERROR();
  
  // Set the start terms
  uint64_t last_char = num_chars - 1;
  uint64_t preprocess_blocks_needed = (workers_needed + THREADS_PER_BLOCK_STRINGS - 1)/THREADS_PER_BLOCK_STRINGS;
  SIRIUS_LOG_DEBUG("Sirius running preprocessing for {} workers with {} strings and {} chars", workers_needed, num_strings, num_chars);

  auto preprocessing_start = std::chrono::high_resolution_clock::now();
  determine_start_kernel<uint64_t><<<preprocess_blocks_needed, THREADS_PER_BLOCK_STRINGS>>>(str_indices, num_strings, d_worker_start_term, 
            workers_needed, CHUNK_SIZE, last_char);
  hipDeviceSynchronize();
  auto preprocessing_end = std::chrono::high_resolution_clock::now();
  int preprocessing_time_us = std::chrono::duration_cast<std::chrono::microseconds>(preprocessing_end - preprocessing_start).count();

  auto str_match_start = std::chrono::high_resolution_clock::now();
  uint64_t block_sub_chunk_size = (CHUNK_SIZE + THREADS_PER_BLOCK_STRINGS - 1)/THREADS_PER_BLOCK_STRINGS;
  single_term_kmp_kernel<uint64_t><<<workers_needed, THREADS_PER_BLOCK_STRINGS>>>(char_data, str_indices, d_kmp_automato, d_worker_start_term, 
    d_answers, match_length, workers_needed, CHUNK_SIZE, block_sub_chunk_size, last_char, num_strings);
  hipDeviceSynchronize();
  auto str_match_end = std::chrono::high_resolution_clock::now();
  int str_match_time_us = std::chrono::duration_cast<std::chrono::microseconds>(str_match_end - str_match_start).count();
  CHECK_ERROR();

  hipMemset(count, 0, sizeof(uint64_t));
  compact_valid_rows<BLOCK_THREADS, ITEMS_PER_THREAD><<<((num_strings + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD)), BLOCK_THREADS>>>(d_answers, row_id, (unsigned long long*) count, num_strings, 1, not_equal);

  // Record the number of valid strings
  uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
  hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
  CHECK_ERROR();
  row_id = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);

  hipMemset(count, 0, sizeof(uint64_t));
  compact_valid_rows<BLOCK_THREADS, ITEMS_PER_THREAD><<<((num_strings + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD)), BLOCK_THREADS>>>(d_answers, row_id, (unsigned long long*) count, num_strings, 0, not_equal);

  // Check there are no errors
  CHECK_ERROR();

  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_match_str), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_kmp_automato), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_worker_start_term), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_answers), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(count), 0);
  count = h_count;
  SIRIUS_LOG_DEBUG("String Matching Result Count = {}", h_count[0]);

  STOP_TIMER();
}

//--------------------------------------------------//
// Multi-Term String Matching
//--------------------------------------------------//
template<typename IdxT>
__global__ void multi_term_kmp_kernel(const char* char_data, const IdxT* indices, const int* kmp_automato, IdxT* worker_start_term, 
  IdxT* curr_term_answer, IdxT* prev_term_answer, bool* found_term, int pattern_size, IdxT num_workers, IdxT chunk_size, IdxT sub_chunk_size, 
  IdxT last_char, IdxT num_strings) {
    
    // See if have any work to do
    auto chunk_id = blockIdx.x;
    if (chunk_id >= num_workers) return;

    const auto curr_chunk_start = min(chunk_id * chunk_size, last_char);
    const auto curr_chunk_end = min(curr_chunk_start + chunk_size + pattern_size, last_char);
    const auto curr_sub_chunk_start = min(curr_chunk_start + threadIdx.x * sub_chunk_size, curr_chunk_end);
    const auto curr_sub_chunk_end = min(curr_sub_chunk_start + sub_chunk_size + pattern_size, curr_chunk_end);

    // Determine the subchunk that the current string is going to be working on
    auto curr_term = worker_start_term[chunk_id];
    while (curr_term < num_strings && (curr_sub_chunk_start < indices[curr_term] || curr_sub_chunk_start >= indices[curr_term + 1])) {
      curr_term++;
    }
    auto curr_term_end = indices[curr_term + 1];

    // Perform the actual string matching
    int j = 0; int curr_idx = 0; 
    #pragma unroll
    for(int i = curr_sub_chunk_start; i <= curr_sub_chunk_end; i++) {
      // See if we need to switch to a new term
      if(i >= curr_term_end) {
          curr_term = curr_term + 1;
          curr_term_end = indices[curr_term + 1];
          j = 0; // Reset because we are at the start of the string
      }

      curr_idx = (int) char_data[i] + CHAR_INCREMENT;
      j = kmp_automato[j * CHARS_IN_BYTE + curr_idx];

      // Record that we have a hit
      if(j >= pattern_size) {
        // Only write the result if we current match index is > than the lowest match index for the previous term
        if(i >= prev_term_answer[curr_term]) {
          found_term[curr_term] = true;
          cuda::atomic_ref<IdxT, cuda::thread_scope_device> curr_term_answer_ref(curr_term_answer[curr_term]);
          curr_term_answer_ref.fetch_min(i + pattern_size, cuda::std::memory_order_relaxed);
        }

        j = 0;
      }
    }
}

// Instantiations
template __global__ void multi_term_kmp_kernel<uint64_t>(const char* char_data,
                                                         const uint64_t* indices,
                                                         const int* kmp_automato,
                                                         uint64_t* worker_start_term,
                                                         uint64_t* curr_term_answer,
                                                         uint64_t* prev_term_answer,
                                                         bool* found_term,
                                                         int pattern_size,
                                                         uint64_t num_workers,
                                                         uint64_t chunk_size,
                                                         uint64_t sub_chunk_size,
                                                         uint64_t last_char,
                                                         uint64_t num_strings);
template __global__ void multi_term_kmp_kernel<int64_t>(const char* char_data,
                                                        const int64_t* indices,
                                                        const int* kmp_automato,
                                                        int64_t* worker_start_term,
                                                        int64_t* curr_term_answer,
                                                        int64_t* prev_term_answer,
                                                        bool* found_term,
                                                        int pattern_size,
                                                        int64_t num_workers,
                                                        int64_t chunk_size,
                                                        int64_t sub_chunk_size,
                                                        int64_t last_char,
                                                        int64_t num_strings);

template<typename IdxT>
__global__ void initialize_term_answers(IdxT* curr_term_answer, IdxT num_chars, IdxT num_strings) {
    const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < num_strings) {
      curr_term_answer[tid] = num_chars;
    } 
}

// Instantiations
template __global__ void initialize_term_answers<uint64_t>(uint64_t* curr_term_answer,
                                                           uint64_t num_chars,
                                                           uint64_t num_strings);
template __global__ void
initialize_term_answers<int64_t>(int64_t* curr_term_answer, int64_t num_chars, int64_t num_strings);

void MultiStringMatching(char* char_data, uint64_t* str_indices, std::vector<std::string> all_terms,
       uint64_t* &row_id, uint64_t* &count, uint64_t num_chars, uint64_t num_strings, int not_equal) {
  CHECK_ERROR();
  GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
  if (num_strings == 0) {
    SIRIUS_LOG_DEBUG("Input size is 0");
    uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
    h_count[0] = 0;
    count = h_count;
    return;
  }
  
    SETUP_TIMING();
    START_TIMER();
  SIRIUS_LOG_DEBUG("Launching multi term string matching kernel");
  // Get the data from the metadata
  uint64_t workers_needed = (num_chars + CHUNK_SIZE - 1)/CHUNK_SIZE;

  // Create the automato for each term
  int num_terms = all_terms.size();
  int** all_terms_automato = gpuBufferManager->customCudaHostAlloc<int*>(num_terms);
  for(int i = 0; i < num_terms; i++) {
    std::string curr_term = all_terms[i];
    const int match_length = curr_term.size();
    const char* match_char = curr_term.c_str();
    int kmp_automato_size = match_length * CHARS_IN_BYTE;
    int* kmp_automato = gpuBufferManager->customCudaHostAlloc<int>(kmp_automato_size);
    std::memset(kmp_automato, 0, kmp_automato_size * sizeof(int));

    // Create the automato for this term
    int first_idx = (int) match_char[0] + CHAR_INCREMENT;
    kmp_automato[first_idx] = 1;
    for(int X = 0, j = 1; j < match_length; j++) {
      int curr_idx = (int) match_char[j] + CHAR_INCREMENT;
      for(int c = 0; c < CHARS_IN_BYTE; c++) {
        kmp_automato[j * CHARS_IN_BYTE + c] = kmp_automato[X * CHARS_IN_BYTE + c];
      }
      kmp_automato[j * CHARS_IN_BYTE + curr_idx] = j + 1;
      X = kmp_automato[X * CHARS_IN_BYTE + curr_idx];
    }

    // Save the automato for this term
    all_terms_automato[i] = kmp_automato;
  }

  // Allocate the buffers on the GPU 
  count = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
  uint64_t* d_worker_start_term = gpuBufferManager->customCudaMalloc<uint64_t>(workers_needed, 0, 0);
  uint64_t* d_prev_term_answers = gpuBufferManager->customCudaMalloc<uint64_t>(num_strings, 0, 0);
  uint64_t* d_answer_idxs = gpuBufferManager->customCudaMalloc<uint64_t>(num_strings, 0, 0);
  hipMemset(d_answer_idxs, 0, num_strings * sizeof(uint64_t));
  bool* d_found_answer = reinterpret_cast<bool*> (gpuBufferManager->customCudaMalloc<uint8_t>(num_strings, 0, 0));
  hipMemset(d_found_answer, 0, num_strings * sizeof(bool));
  // uint64_t* d_matching_rows = gpuBufferManager->customCudaMalloc<uint64_t>(num_strings, 0, 0);

  // Create buffer for each automato
  int** d_all_automatos = gpuBufferManager->customCudaHostAlloc<int*>(num_terms);
  for(int i = 0; i < num_terms; i++) {
    int kmp_automato_size = all_terms[i].size() * CHARS_IN_BYTE;
    d_all_automatos[i] = gpuBufferManager->customCudaMalloc<int>(kmp_automato_size * sizeof(int), 0, 0);
  }

  // Copy over the necessary data 
  hipMemcpy(d_prev_term_answers, str_indices, num_strings * sizeof(uint64_t), hipMemcpyDeviceToDevice);
  for(int i = 0; i < num_terms; i++) {
    int kmp_automato_size = all_terms[i].size() * CHARS_IN_BYTE;
    hipMemcpy(d_all_automatos[i], all_terms_automato[i], kmp_automato_size * sizeof(int), hipMemcpyHostToDevice);
  }

  // Initialize the other buffers
  // hipMemset(d_matching_rows, 0, num_strings * sizeof(uint64_t));
  CHECK_ERROR();

  // Determine the start offset for each kernel
  uint64_t last_char = num_chars - 1;
  uint64_t kernel_block_needed = (workers_needed + THREADS_PER_BLOCK_STRINGS - 1)/THREADS_PER_BLOCK_STRINGS;
  uint64_t block_sub_chunk_size = (CHUNK_SIZE + THREADS_PER_BLOCK_STRINGS - 1)/THREADS_PER_BLOCK_STRINGS;
  determine_start_kernel<uint64_t><<<kernel_block_needed, THREADS_PER_BLOCK_STRINGS>>>(str_indices, num_strings, d_worker_start_term, 
            workers_needed, CHUNK_SIZE, last_char);
  CHECK_ERROR();
  
  // Perform the string matching term by term
  uint64_t preprocess_num_blocks = (num_strings + THREADS_PER_BLOCK_STRINGS - 1)/THREADS_PER_BLOCK_STRINGS;
  for(int i = 0; i < num_terms; i++) {
    // Determine the current terms variables
    int curr_term_length = all_terms[i].size();
    int* curr_term_automato = d_all_automatos[i];

    // Perform pre processing
    hipMemset(d_found_answer, 0, num_strings * sizeof(bool));
    initialize_term_answers<uint64_t><<<preprocess_num_blocks, THREADS_PER_BLOCK_STRINGS>>>(d_answer_idxs, num_chars, num_strings);
    CHECK_ERROR();

    // Run the search
    multi_term_kmp_kernel<uint64_t><<<workers_needed, THREADS_PER_BLOCK_STRINGS>>>(char_data, str_indices, curr_term_automato, d_worker_start_term, 
      d_answer_idxs, d_prev_term_answers, d_found_answer, curr_term_length, workers_needed, CHUNK_SIZE, block_sub_chunk_size, 
      last_char, num_strings);
    CHECK_ERROR();

    // If there are future terms, the make the current answer the prev term answers
    if(i < (num_terms - 1)) {
      uint64_t* temp_ptr = d_answer_idxs;
      d_answer_idxs = d_prev_term_answers;
      d_prev_term_answers = temp_ptr;
    }
  }

  hipMemset(count, 0, sizeof(uint64_t));
  compact_valid_rows<BLOCK_THREADS, ITEMS_PER_THREAD><<<((num_strings + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD)), BLOCK_THREADS>>>(d_found_answer, row_id, (unsigned long long*) count, num_strings, 1, not_equal);

  // Record the number of valid strings
  uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
  hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
  CHECK_ERROR();
  row_id = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
  hipMemset(count, 0, sizeof(uint64_t));

  compact_valid_rows<BLOCK_THREADS, ITEMS_PER_THREAD><<<((num_strings + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD)), BLOCK_THREADS>>>(d_found_answer, row_id, (unsigned long long*) count, num_strings, 0, not_equal);

  // Check there are no errors
  CHECK_ERROR();

  //free the memory
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_worker_start_term), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_prev_term_answers), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>( d_answer_idxs), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_found_answer), 0);
  for(int i = 0; i < num_terms; i++) {
    gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_all_automatos[i]), 0);
  }
  SIRIUS_LOG_DEBUG("Multi String Matching Result Count = {}", h_count[0]);
  STOP_TIMER();

  count = h_count;
}

//--------------------------------------------------//
// Prefix Matching
//--------------------------------------------------//
template<typename IdxT>
__global__ void prefix_kernel(const char* char_data, IdxT num_chars, const IdxT* str_indices, IdxT num_strings, const char* prefix_chars, 
  IdxT num_prefix_chars, bool* results) {
  const IdxT start_idx = threadIdx.x + blockIdx.x * blockDim.x;
  const IdxT tile_size = gridDim.x * blockDim.x;
  for(IdxT i = start_idx; i < num_strings; i += tile_size) {
    // First get the current strings details and check its length
    IdxT start_offset = str_indices[i]; 
    IdxT end_offset = str_indices[i + 1];
    IdxT curr_str_length = end_offset - start_offset;
    if(curr_str_length < num_prefix_chars) {
      results[i] = false;
      continue;
    }
    const char* curr_str_chars = char_data + start_offset;
    // Now actually compare the initial chars
    bool is_valid = true;
    for(IdxT j = 0; j < num_prefix_chars; j++) {
      if(curr_str_chars[j] != prefix_chars[j]) {
        is_valid = false;
        break;
      }
    }
    results[i] = is_valid;
  }
}

// Instantiations
template __global__ void prefix_kernel<uint64_t>(const char* char_data,
                                                 uint64_t num_chars,
                                                 const uint64_t* str_indices,
                                                 uint64_t num_strings,
                                                 const char* prefix_chars,
                                                 uint64_t num_prefix_chars,
                                                 bool* results);
template __global__ void prefix_kernel<int64_t>(const char* char_data,
                                                int64_t num_chars,
                                                const int64_t* str_indices,
                                                int64_t num_strings,
                                                const char* prefix_chars,
                                                int64_t num_prefix_chars,
                                                bool* results);

void PrefixMatching(char* char_data, uint64_t* str_indices, std::string match_prefix, uint64_t* &row_id, uint64_t* &count, 
  uint64_t num_chars, uint64_t num_strings, int not_equal) {

  // Allocate the necesary buffers on the GPU
  GPUBufferManager* gpuBufferManager = &(GPUBufferManager::GetInstance());
  if (num_strings == 0) {
    SIRIUS_LOG_DEBUG("Input size is 0");
    uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
    h_count[0] = 0;
    count = h_count;
    return;
  }

    SETUP_TIMING();
    START_TIMER();
    SIRIUS_LOG_DEBUG("Launching Prefix Matching kernel");

  count = gpuBufferManager->customCudaMalloc<uint64_t>(1, 0, 0);
  uint64_t num_prefix_chars = match_prefix.length();
  char* d_prefix_chars = gpuBufferManager->customCudaMalloc<char>(num_prefix_chars, 0, 0);
  hipMemcpy(d_prefix_chars, match_prefix.c_str(), num_prefix_chars * sizeof(char), hipMemcpyHostToDevice);
  bool* d_results = gpuBufferManager->customCudaMalloc<bool>(num_strings, 0, 0);
  hipMemset(d_results, 0, num_strings * sizeof(bool));

  // Run the kernel
  uint64_t items_per_block = BLOCK_THREADS * ITEMS_PER_THREAD;
  uint64_t num_blocks = (num_strings + items_per_block - 1)/items_per_block;
  prefix_kernel<<<num_blocks, BLOCK_THREADS>>>(char_data, num_chars, str_indices, num_strings, d_prefix_chars, num_prefix_chars, d_results);
  hipDeviceSynchronize();
  CHECK_ERROR();

  hipMemset(count, 0, sizeof(uint64_t));
  compact_valid_rows<BLOCK_THREADS, ITEMS_PER_THREAD><<<((num_strings + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD)), BLOCK_THREADS>>>(d_results, row_id, (unsigned long long*) count, num_strings, 1, not_equal);

  // Record the number of valid strings
  uint64_t* h_count = gpuBufferManager->customCudaHostAlloc<uint64_t>(1);
  hipMemcpy(h_count, count, sizeof(uint64_t), hipMemcpyDeviceToHost);
  CHECK_ERROR();
  row_id = gpuBufferManager->customCudaMalloc<uint64_t>(h_count[0], 0, 0);
  hipMemset(count, 0, sizeof(uint64_t));

  compact_valid_rows<BLOCK_THREADS, ITEMS_PER_THREAD><<<((num_strings + BLOCK_THREADS * ITEMS_PER_THREAD - 1)/(BLOCK_THREADS * ITEMS_PER_THREAD)), BLOCK_THREADS>>>(d_results, row_id, (unsigned long long*) count, num_strings, 0, not_equal);

  // Check there are no errors
  hipDeviceSynchronize();
  CHECK_ERROR();

  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_prefix_chars), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(d_results), 0);
  gpuBufferManager->customCudaFree(reinterpret_cast<uint8_t*>(count), 0);

  count = h_count;
  SIRIUS_LOG_DEBUG("PrefixMatching Result Count {}", h_count[0]);
  STOP_TIMER();
}

//--------------------------------------------------//
// String Matching for CuDF Compatibility
//--------------------------------------------------//
// This is a replication of the above functions for compatibility with CuDF
// The key points are 1) use cudf::size_type instead of uint64_t
//                    2) manage memory with rmm::device_uvector, so ownership can be transferred
//                       to cudf::columns
//                    3) emit a boolean column instead of row ids

// Macros to simplify kernel launch syntax
#define LAUNCH_KERNEL_DIV(K, T, N, B, S) K<T><<<cuda::ceil_div((N), (B)), (B), 0, (S)>>>
#define LAUNCH_KERNEL_DIRECT(K, T, N, B, S) K<T><<<(N), (B), 0, (S)>>>

//----------String Matching----------//
std::unique_ptr<cudf::column> DoStringMatching(const char* input_data,
                                               cudf::size_type input_count,
                                               const int64_t* input_offsets,
                                               int64_t byte_count,
                                               const std::string& match_string,
                                               rmm::device_async_resource_ref mr,
                                               rmm::cuda_stream_view stream)
{
  static_assert(std::is_same_v<int32_t, cudf::size_type>); // Sanity check

  // Compute the automato for this string
  const auto match_length      = static_cast<int32_t>(match_string.size());
  const auto* match_char       = match_string.c_str();
  const auto kmp_automato_size = match_length * CHARS_IN_BYTE;
  std::vector<int32_t> kmp_automato(kmp_automato_size, 0);
  const auto first_idx    = static_cast<int32_t>(match_char[0]) + CHAR_INCREMENT;
  kmp_automato[first_idx] = 1;
  for (int32_t X = 0, j = 1; j < match_length; j++)
  {
    const auto curr_idx = static_cast<int32_t>(match_char[j]) + CHAR_INCREMENT;

    // Copy over the chars from the previous automato
    for (int32_t c = 0; c < CHARS_IN_BYTE; c++)
    {
      kmp_automato[j * CHARS_IN_BYTE + c] = kmp_automato[X * CHARS_IN_BYTE + c];
    }
    kmp_automato[j * CHARS_IN_BYTE + curr_idx] = j + 1;
    X                                          = kmp_automato[X * CHARS_IN_BYTE + curr_idx];
  }

  // Copy match string to device memory
  rmm::device_uvector<char> d_match_string(match_length, stream, mr);
  CUDF_CUDA_TRY(hipMemcpyAsync(d_match_string.data(),
                                match_string.data(),
                                match_length * sizeof(char),
                                hipMemcpyHostToDevice,
                                stream));

  // Copy automato to device memory
  rmm::device_uvector<int32_t> d_kmp_automato(kmp_automato_size, stream, mr);
  CUDF_CUDA_TRY(hipMemcpyAsync(d_kmp_automato.data(),
                                kmp_automato.data(),
                                kmp_automato_size * sizeof(int32_t),
                                hipMemcpyHostToDevice,
                                stream));

  // Allocate start terms memory and the boolean output buffer
  const auto workers_needed = cuda::ceil_div(byte_count, static_cast<int64_t>(CHUNK_SIZE));
  rmm::device_uvector<int64_t> d_worker_start_term(workers_needed, stream, mr);
  rmm::device_uvector<bool> output(input_count, stream, mr);

  // Initialize the output buffer to false
  CUDF_CUDA_TRY(hipMemsetAsync(output.data(), 0, input_count * sizeof(bool), stream));

  // Launch kernel to determine the start offset for each worker
  LAUNCH_KERNEL_DIV(determine_start_kernel,
                    int64_t,
                    workers_needed,
                    static_cast<int64_t>(THREADS_PER_BLOCK_STRINGS),
                    stream)
  (input_offsets,
   input_count,
   d_worker_start_term.data(),
   workers_needed,
   CHUNK_SIZE,
   byte_count - 1);

  // Launch KMP kernel
  LAUNCH_KERNEL_DIRECT(single_term_kmp_kernel,
                       int64_t,
                       workers_needed,
                       THREADS_PER_BLOCK_STRINGS,
                       stream)
  (input_data,
   input_offsets,
   d_kmp_automato.data(),
   d_worker_start_term.data(),
   output.data(),
   match_length,
   workers_needed,
   CHUNK_SIZE,
   cuda::ceil_div(CHUNK_SIZE, THREADS_PER_BLOCK_STRINGS),
   byte_count - 1,
   input_count);

  // Return a boolean cudf::column
  return std::make_unique<cudf::column>(std::move(output), rmm::device_buffer(0, stream, mr), 0);
}

//----------Multi-Term String Matching----------//
std::unique_ptr<cudf::column> DoMultiStringMatching(const char* input_data,
                                                    cudf::size_type input_count,
                                                    const int64_t* input_offsets,
                                                    int64_t byte_count,
                                                    const std::vector<std::string>& match_strings,
                                                    rmm::device_async_resource_ref mr,
                                                    rmm::cuda_stream_view stream)
{
  static_assert(std::is_same_v<int32_t, cudf::size_type>); // Sanity check

  // Compute the automato for each term
  std::vector<rmm::device_uvector<int32_t>> d_kmp_automatos;
  for (const auto& match_string : match_strings)
  {
    const auto match_length      = static_cast<int32_t>(match_string.size());
    const auto* match_char       = match_string.c_str();
    const auto kmp_automato_size = match_length * CHARS_IN_BYTE;
    std::vector<int32_t> kmp_automato(kmp_automato_size, 0);
    const auto first_idx    = static_cast<int32_t>(match_char[0]) + CHAR_INCREMENT;
    kmp_automato[first_idx] = 1;
    for (int32_t X = 0, j = 1; j < match_length; j++)
    {
      const auto curr_idx = static_cast<int32_t>(match_char[j]) + CHAR_INCREMENT;

      // Copy over the chars from the previous automato
      for (int32_t c = 0; c < CHARS_IN_BYTE; c++)
      {
        kmp_automato[j * CHARS_IN_BYTE + c] = kmp_automato[X * CHARS_IN_BYTE + c];
      }
      kmp_automato[j * CHARS_IN_BYTE + curr_idx] = j + 1;
      X                                          = kmp_automato[X * CHARS_IN_BYTE + curr_idx];
    }

    // Copy automato to device memory
    d_kmp_automatos.emplace_back(kmp_automato_size, stream, mr);
    CUDF_CUDA_TRY(hipMemcpyAsync(d_kmp_automatos.back().data(),
                                  kmp_automato.data(),
                                  kmp_automato_size * sizeof(int32_t),
                                  hipMemcpyHostToDevice,
                                  stream));
  }

  // Allocate start terms memory, rotating answer indices, and the boolean output buffer
  const auto workers_needed = cuda::ceil_div(byte_count, static_cast<int64_t>(CHUNK_SIZE));
  rmm::device_uvector<int64_t> d_worker_start_term(workers_needed, stream, mr);
  rmm::device_uvector<int64_t> d_answer_idxs(input_count, stream, mr);
  rmm::device_uvector<int64_t> d_prev_answer_idxs(input_count, stream, mr);
  rmm::device_uvector<bool> output(input_count, stream, mr);

  // Initialize answer indices to zero, and copy offsets to previous answer indices
  CUDF_CUDA_TRY(hipMemsetAsync(d_answer_idxs.data(), 0, input_count * sizeof(int64_t), stream));
  CUDF_CUDA_TRY(hipMemcpyAsync(d_prev_answer_idxs.data(),
                                input_offsets,
                                input_count * sizeof(int64_t),
                                hipMemcpyDeviceToDevice,
                                stream));

  // Launch kernel to determine the start offset for each worker
  LAUNCH_KERNEL_DIV(determine_start_kernel,
                    int64_t,
                    workers_needed,
                    static_cast<int64_t>(THREADS_PER_BLOCK_STRINGS),
                    stream)
  (input_offsets,
   input_count,
   d_worker_start_term.data(),
   workers_needed,
   CHUNK_SIZE,
   byte_count - 1);

  // Perform the string matching for each term
  auto* answer_idxs_ptr      = d_answer_idxs.data();
  auto* prev_answer_idxs_ptr = d_prev_answer_idxs.data();
  for (int32_t i = 0; i < match_strings.size(); i++)
  {
    const auto curr_term_length    = static_cast<int32_t>(match_strings[i].size());
    const auto* curr_term_automato = d_kmp_automatos[i].data();

    // Preprocessing
    CUDF_CUDA_TRY(hipMemsetAsync(output.data(), 0, input_count * sizeof(bool), stream));
    LAUNCH_KERNEL_DIV(initialize_term_answers,
                      int64_t,
                      input_count,
                      THREADS_PER_BLOCK_STRINGS,
                      stream)
    (answer_idxs_ptr, byte_count, input_count);

    // Launch the KMP kernel for the current term
    LAUNCH_KERNEL_DIRECT(multi_term_kmp_kernel,
                         int64_t,
                         workers_needed,
                         THREADS_PER_BLOCK_STRINGS,
                         stream)
    (input_data,
     input_offsets,
     curr_term_automato,
     d_worker_start_term.data(),
     answer_idxs_ptr,
     prev_answer_idxs_ptr,
     output.data(),
     curr_term_length,
     workers_needed,
     CHUNK_SIZE,
     cuda::ceil_div(CHUNK_SIZE, THREADS_PER_BLOCK_STRINGS),
     byte_count - 1,
     input_count);

    // If there are future terms, swap the answer indices
    if (i < (match_strings.size() - 1))
    {
      // Swap the answer indices for the next term
      std::swap(answer_idxs_ptr, prev_answer_idxs_ptr);
    }
  }

  // Return a boolean cudf::column
  return std::make_unique<cudf::column>(std::move(output), rmm::device_buffer(0, stream, mr), 0);
}

//----------Prefix Matching----------//
std::unique_ptr<cudf::column> DoPrefixMatching(const char* input_data,
                                               cudf::size_type input_count,
                                               const int64_t* input_offsets,
                                               int64_t byte_count,
                                               const std::string& match_prefix,
                                               rmm::device_async_resource_ref mr,
                                               rmm::cuda_stream_view stream)
{
  static_assert(std::is_same_v<int32_t, cudf::size_type>); // Sanity check

  // Copy prefix string to device memory
  const auto prefix_byte_count = static_cast<int32_t>(match_prefix.size());
  rmm::device_uvector<char> d_match_prefix(prefix_byte_count, stream, mr);
  CUDF_CUDA_TRY(hipMemcpyAsync(d_match_prefix.data(),
                                match_prefix.data(),
                                prefix_byte_count,
                                hipMemcpyHostToDevice,
                                stream));

  // Allocate boolean output buffer
  rmm::device_uvector<bool> output(input_count, stream, mr);

  // Launch kernel to perform prefix matching
  LAUNCH_KERNEL_DIV(prefix_kernel, int64_t, input_count, BLOCK_THREADS, stream)
  (input_data,
   byte_count,
   input_offsets,
   input_count,
   d_match_prefix.data(),
   prefix_byte_count,
   output.data());

  // Return a cudf::column
  return std::make_unique<cudf::column>(std::move(output), rmm::device_buffer(0, stream, mr), 0);
}

#undef LAUNCH_KERNEL_DIV
#undef LAUNCH_KERNEL_DIRECT

} // namespace duckdb
