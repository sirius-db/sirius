/*
 * Copyright 2025, Sirius Contributors.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "expression_executor/gpu_dispatcher.hpp"
#include "gpu_buffer_manager.hpp"
#include <hipcub/hipcub.hpp>
#include <tuple>

namespace duckdb
{
namespace sirius
{

//----------Select----------//
std::tuple<uint64_t*, uint64_t> GpuDispatcher::DispatchSelect(const cudf::column_view& bitmap,
                                                              rmm::device_async_resource_ref mr,
                                                              rmm::cuda_stream_view stream)
{
  // The row ids are owned by the query executor and so must be managed by the buffer manager
  auto* gpu_buffer_manager = &GPUBufferManager::GetInstance();
  uint64_t* row_ids = gpu_buffer_manager->customCudaMalloc<uint64_t>(bitmap.size(), 0, false);
  rmm::device_scalar<uint64_t> d_num_selected(0, stream, mr);

  size_t temp_storage_bytes = 0;
  uint64_t num_selected     = 0;
  hipcub::DeviceSelect::Flagged(nullptr,
                             temp_storage_bytes,
                             thrust::make_counting_iterator<uint64_t>(0),
                             bitmap.data<bool>(),
                             row_ids,
                             d_num_selected.data(),
                             bitmap.size(),
                             stream);
  rmm::device_buffer temp_storage(temp_storage_bytes, stream, mr);
  hipcub::DeviceSelect::Flagged(temp_storage.data(),
                             temp_storage_bytes,
                             thrust::make_counting_iterator<uint64_t>(0),
                             bitmap.data<bool>(),
                             row_ids,
                             d_num_selected.data(),
                             bitmap.size(),
                             stream);
  num_selected = d_num_selected.value(stream);
  return std::make_tuple(row_ids, num_selected);
}

} // namespace sirius
} // namespace duckdb