#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "communication.hpp"
#include "gpu_buffer_manager.hpp"
#include "operator/cuda_helper.cuh"

namespace duckdb {

template void
callCudaMemcpyHostToDevice<int>(int* dest, int* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<uint64_t>(uint64_t* dest, uint64_t* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<float>(float* dest, float* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<double>(double* dest, double* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<uint8_t>(uint8_t* dest, uint8_t* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<char>(char* dest, char* src, size_t size, int gpu);

template void
callCudaMemcpyHostToDevice<bool>(bool* dest, bool* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<int>(int* dest, int* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<uint64_t>(uint64_t* dest, uint64_t* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<float>(float* dest, float* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<double>(double* dest, double* src, size_t size, int gpu);

template void
callCudaMemcpyDeviceToHost<uint8_t>(uint8_t* dest, uint8_t* src, size_t size, int gpu);

template <typename T> 
void callCudaMemcpyHostToDevice(T* dest, T* src, size_t size, int gpu) {
    printf("callCudaMemcpyHostToDevice Send data to GPU\n");
    hipSetDevice(gpu);
    gpuErrchk(hipMemcpy(dest, src, size * sizeof(T), hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());
    hipSetDevice(0);
    printf("callCudaMemcpyHostToDevice Done sending data to GPU\n");
}

template <typename T> 
void callCudaMemcpyDeviceToHost(T* dest, T* src, size_t size, int gpu) {
    printf("Send data to CPU\n");
    hipSetDevice(gpu);
    gpuErrchk(hipMemcpy(dest, src, size * sizeof(T), hipMemcpyDeviceToHost));
    CHECK_ERROR();
    gpuErrchk(hipDeviceSynchronize());
    hipSetDevice(0);
    printf("Done sending data to CPU\n");
}

// Define the host function that launches the CUDA kernel
int* sendDataToGPU(int* data, int size) {
    printf("sendDataToGPU Send data to GPU\n");
    // use cudamemcpy
    int* target;
    hipMalloc((void**) &target, size * sizeof(int));
    hipMemcpy(target, data, size * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    printf("sendDataToGPU Done sending data to GPU\n");
    return target;
}

} // namespace duckdb